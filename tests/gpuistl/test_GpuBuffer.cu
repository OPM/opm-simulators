/*
  Copyright 2024 SINTEF AS

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestGpuBuffer

#include <boost/test/unit_test.hpp>
#include <hip/hip_runtime.h>

#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>

#include <array>
#include <algorithm>
#include <type_traits>

BOOST_AUTO_TEST_CASE(TestMakeView)
{
    // test that we can create buffers and make views of the buffers using the pointer constructor
    auto buf = std::vector<int>({1, 2, 3, 4, 5, 6});
    const auto gpubuf = ::Opm::gpuistl::GpuBuffer<int>(buf);
    auto gpuview = ::Opm::gpuistl::GpuView<int>(buf.data(), buf.size());
    bool gpuBufCreatedView = std::is_same<::Opm::gpuistl::GpuView<int>, decltype(gpuview)>::value;

    BOOST_CHECK(gpuBufCreatedView);

    // test that we can make views of buffers by using the GpuBuffer constructor
    auto gpuview2 = ::Opm::gpuistl::make_view(gpubuf);
    bool gpuBufCreatedView2 = std::is_same<::Opm::gpuistl::GpuView<const int>, decltype(gpuview2)>::value;

    BOOST_CHECK(gpuBufCreatedView2);

    // check that we retrieve the same values when pulling the data back to the cpu as a vector
    auto gpuBufOnCpu = gpubuf.asStdVector();
    BOOST_CHECK_EQUAL_COLLECTIONS(gpuBufOnCpu.begin(), gpuBufOnCpu.end(), buf.begin(), buf.end());
}
