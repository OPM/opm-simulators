#include "hip/hip_runtime.h"
/*
  Copyright 2024 SINTEF AS

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestGpuView

#include <boost/test/unit_test.hpp>
#include <hip/hip_runtime.h>
#include <dune/common/fvector.hh>
#include <dune/istl/bvector.hh>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <random>
#include <array>
#include <algorithm>
#include <type_traits>

using GpuViewDouble = ::Opm::gpuistl::GpuView<double>;
using GpuBufferDouble = ::Opm::gpuistl::GpuBuffer<double>;

__global__ void useGpuViewOnGPU(GpuViewDouble a, GpuViewDouble b){
    b[0] = a.front();
    b[1] = a.back();
    b[2] = *a.begin();
    b[3] = *(--a.end());

    a[0] = a[2];
}

BOOST_AUTO_TEST_CASE(TestCreationAndIndexing)
{
    // A simple test to check that we can move data to and from the GPU
    auto cpubuffer = std::vector<double>({1.0, 2.0, 42.0, 59.9451743, 10.7132692});
    auto cubuffer = GpuBufferDouble(cpubuffer);
    auto gpuview = GpuViewDouble(cubuffer.data(), cubuffer.size());
    const auto const_gpuview = GpuViewDouble(cubuffer.data(), cubuffer.size());

    auto stdVecOfGpuView = gpuview.asStdVector();
    auto const_stdVecOfGpuView = gpuview.asStdVector();

    BOOST_CHECK_EQUAL_COLLECTIONS(
        stdVecOfGpuView.begin(), stdVecOfGpuView.end(), cpubuffer.begin(), cpubuffer.end());
    BOOST_CHECK_EQUAL_COLLECTIONS(
        stdVecOfGpuView.begin(), stdVecOfGpuView.end(), const_stdVecOfGpuView.begin(), const_stdVecOfGpuView.end());
}

BOOST_AUTO_TEST_CASE(TestGpuViewOnCPUTypes)
{
    auto buf = std::vector<double>({1.0, 2.0, 42.0, 59.9451743, 10.7132692});
    auto cpuview = GpuViewDouble(buf.data(), buf.size());
    const auto const_cpuview = GpuViewDouble(buf.data(), buf.size());

    // check that indexing a const view produces a value
    bool correct_type_of_const_cpu_front = std::is_same_v<double, decltype(const_cpuview.front())>;
    bool correct_type_of_const_cpu_back = std::is_same_v<double, decltype(const_cpuview.back())>;

    BOOST_CHECK(correct_type_of_const_cpu_front);
    BOOST_CHECK(correct_type_of_const_cpu_back);

    // check that the values are correct
    BOOST_CHECK(const_cpuview.front() == buf.front());
    BOOST_CHECK(const_cpuview.back() == buf.back());
}

BOOST_AUTO_TEST_CASE(TestGpuViewOnCPUWithSTLIteratorAlgorithm)
{
    auto buf = std::vector<double>({1.0, 2.0, 42.0, 59.9451743, 10.7132692});
    auto cpuview = GpuViewDouble(buf.data(), buf.size());
    std::sort(buf.begin(), buf.end());
    BOOST_CHECK(42.0 == cpuview[3]);
}

BOOST_AUTO_TEST_CASE(TestGpuViewOnGPU)
{
    auto buf = std::vector<double>({1.0, 2.0, 42.0, 59.9451743, 10.7132692});
    auto cubufA = GpuBufferDouble(buf);
    auto gpuviewA = GpuViewDouble(cubufA.data(), cubufA.size());
    auto cubufB = GpuBufferDouble(4);
    auto gpuviewB = GpuViewDouble(cubufB.data(), cubufB.size());

    useGpuViewOnGPU<<<1,1>>>(gpuviewA, gpuviewB);

    auto vecA = gpuviewA.asStdVector();
    auto vecB = gpuviewB.asStdVector();

    // checks that front/back/begin/end works
    BOOST_CHECK(vecB[0] == buf[0]);
    BOOST_CHECK(vecB[1] == buf[4]);
    BOOST_CHECK(vecB[2] == buf[0]);
    BOOST_CHECK(vecB[3] == buf[4]);

    // checks that view[0] = view[2] works
    BOOST_CHECK(buf[2] == vecA[0]);
}
