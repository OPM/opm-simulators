#include "hip/hip_runtime.h"
#include <config.h>

#define BOOST_TEST_MODULE TestGpuPvt

#include <boost/test/unit_test.hpp>
#include <opm/material/common/MathToolbox.hpp>
#include <opm/material/densead/Evaluation.hpp>
#include <opm/material/densead/Math.hpp>
#include <opm/material/common/UniformTabulated2DFunction.hpp>
#include <opm/material/components/CO2Tables.hpp>
#include <opm/material/components/CO2.hpp>
#include <opm/material/components/SimpleHuDuanH2O.hpp>
#include <opm/material/components/BrineDynamic.hpp>
#include <opm/material/binarycoefficients/Brine_CO2.hpp>
#include <opm/material/fluidsystems/blackoilpvt/Co2GasPvt.hpp>
#include <opm/material/fluidsystems/blackoilpvt/BrineCo2Pvt.hpp>
#include <opm/input/eclipse/EclipseState/Co2StoreConfig.hpp>

#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <opm/simulators/linalg/gpuistl/GpuBuffer.hpp>
#include <opm/simulators/linalg/gpuistl/GpuView.hpp>

#include <hip/hip_runtime.h>
#include <vector>
#include <utility>
#include <cmath>

using Evaluation = Opm::DenseAd::Evaluation<double, 3>;
using GpuB = Opm::gpuistl::GpuBuffer<double>;
using GpuV = Opm::gpuistl::GpuView<double>;

using GpuTab = Opm::UniformTabulated2DFunction<double, GpuV>;

using GpuBufCo2Tables = Opm::CO2Tables<double, GpuB>;
using GpuViewCO2Tables = Opm::CO2Tables<double, GpuV>;
using GpuCO2 = Opm::CO2<double, GpuViewCO2Tables>;

using HuDuan = Opm::SimpleHuDuanH2O<double>;
using BrineDyn = Opm::BrineDynamic<double, HuDuan>;

using CpuBrine_CO2 = Opm::BinaryCoeff::Brine_CO2<double, HuDuan, Opm::CO2<double>>;
using GpuBrine_CO2 = Opm::BinaryCoeff::Brine_CO2<double, HuDuan, GpuCO2>;

using CpuCo2Pvt = Opm::Co2GasPvt<double>;
using GpuBufCo2Pvt = Opm::Co2GasPvt<double, GpuBufCo2Tables, GpuB>;
using GpuViewCo2Pvt = Opm::Co2GasPvt<double, GpuViewCO2Tables, GpuV>;

using CpuBrineCo2Pvt = Opm::BrineCo2Pvt<double>;
using GpuBufBrineCo2Pvt = Opm::BrineCo2Pvt<double, GpuBufCo2Tables, GpuB>;
using GpuViewBrineCo2Pvt = Opm::BrineCo2Pvt<double, GpuViewCO2Tables, GpuV>;

namespace {

/*
    This file contains unit tests for Pvt objects and function on the GPU, with additional helper classes
*/

const double ABS_TOL = 1e-6;

struct Fixture {
    Fixture(){
        temp = Evaluation(290.5);
        pressure = Evaluation(200000.0);

        gpuComputedResultOnCpu = 0.0;

        // move pvt evaluations to gpu
        OPM_GPU_SAFE_CALL(hipMalloc(&gpuTemp, sizeof(Evaluation)));
        OPM_GPU_SAFE_CALL(hipMemcpy(gpuTemp, &temp, sizeof(Evaluation), hipMemcpyHostToDevice));
        OPM_GPU_SAFE_CALL(hipMalloc(&gpuPressure, sizeof(Evaluation)));
        OPM_GPU_SAFE_CALL(hipMemcpy(gpuPressure, &pressure, sizeof(Evaluation), hipMemcpyHostToDevice));
    }
    ~Fixture(){
        OPM_GPU_SAFE_CALL(hipFree(gpuTemp));
        OPM_GPU_SAFE_CALL(hipFree(gpuPressure));
    }

    Evaluation temp; // [K]
    Evaluation pressure; // [Pa]
    Evaluation* gpuTemp; // [K]
    Evaluation* gpuPressure; // [Pa]

    double gpuComputedResultOnCpu;

    Opm::CO2Tables<double, std::vector<double>> co2Tables;
};

// Kernel to evaluate a 2D function on the GPU
__global__ void gpuEvaluateUniformTabulated2DFunction(GpuTab gpuTab, Evaluation* inputX, Evaluation* inputY, double* result) {
    *result = gpuTab.eval(*inputX, *inputY, true).value();
}

// Kernel using a CO2 object on the GPU
__global__ void gpuCO2GasViscosity(GpuViewCO2Tables gpuViewCo2Tables, Evaluation* temp, Evaluation* pressure, double* result) {
    *result = GpuCO2::gasViscosity(gpuViewCo2Tables, *temp, *pressure, true).value();
}

// Kernel using a SimpleHuDuanH20 object on a GPU
__global__ void huDuanLiquidDensity(Evaluation* temp, Evaluation* pressure, double* result) {
    *result = HuDuan::liquidDensity(*temp, *pressure, true).value();
}

// Kernel using a BrineDynamic object on a GPU
__global__ void brineDynamicLiquidEnthalpy(Evaluation* temp, Evaluation* pressure, Evaluation* salinity, double* result) {
    *result = BrineDyn::liquidEnthalpy(*temp, *pressure, *salinity).value();
}

// Kernel using a Brine_CO2 object on a GPU
__global__ void brineCO2GasDiffCoeff(GpuViewCO2Tables co2tables, Evaluation* temp, Evaluation* pressure, double* result) {
    *result = GpuBrine_CO2::gasDiffCoeff<Evaluation, GpuViewCO2Tables>(co2tables, *temp, *pressure, true).value();
}

// Kernel using a Co2GasPvt object on a GPU
__global__ void co2GasPvtInternalEnergy(GpuViewCo2Pvt gpuViewCo2Pvt, Evaluation* temp, Evaluation* pressure, double* result) {
    *result = gpuViewCo2Pvt.internalEnergy(1, *temp, *pressure, Evaluation(0.4), Evaluation(0.0)).value();
}

// Kernel using a BrineCo2Pvt object on a GPU
__global__ void brineCo2PvtInternalEnergy(GpuViewBrineCo2Pvt gpuViewBrineCo2Pvt, Evaluation* temp, Evaluation* pressure, Evaluation* rs, Evaluation* saltConcentration, double* result) {
    *result = gpuViewBrineCo2Pvt.internalEnergy(1, *temp, *pressure, *rs, *saltConcentration).value();
}

// Helper function to launch a kernel and retrieve the result on the CPU to reduce code duplicatoin
template <typename KernelFunc, typename... Args>
double launchKernelAndRetrieveResult(KernelFunc kernel, Args... args) {
    double* resultOnGpu;
    double gpuComputedResultOnCpu;

    // Allocate memory for the result on the GPU
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Launch the kernel
    kernel<<<1, 1>>>(args..., resultOnGpu);

    // Check for any errors in kernel launch
    OPM_GPU_SAFE_CALL(hipPeekAtLastError());
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());

    // Retrieve the result from the GPU to the CPU
    OPM_GPU_SAFE_CALL(hipMemcpy(&gpuComputedResultOnCpu, resultOnGpu, sizeof(double), hipMemcpyDeviceToHost));

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(resultOnGpu));

    return gpuComputedResultOnCpu;
}

bool compareSignificantDigits(double a, double b, int significantDigits) {
    // Handle the case where both values are exactly equal
    if (a == b) {
        return true;
    }

    // Calculate the relative error
    double relativeError = std::abs(a - b) / std::max(std::abs(a), std::abs(b));

    // Compute the number of matching digits
    double digitsMatched = -std::log10(relativeError);

    // Return true if they match the required number of significant digits
    return digitsMatched >= significantDigits;
}

} // END EMPTY NAMESPACE

// Test case for evaluating a tabulated 2D function on both CPU and GPU
BOOST_FIXTURE_TEST_CASE(TestEvaluateUniformTabulated2DFunctionOnGpu, Fixture) {
    // Example tabulated data (2D)
    std::vector<std::vector<double>> tabData = {{1.0, 2.0}, {3.0, 4.0}, {5.0, 6.0}};

    // CPU-side function definition
    Opm::UniformTabulated2DFunction<double> cpuTab(1.0, 6.0, 3, 1.0, 6.0, 2, tabData);

    // Move data to GPU buffer and create a view for GPU operations
    Opm::UniformTabulated2DFunction<double, GpuB> gpuBufTab = Opm::gpuistl::copy_to_gpu<GpuB>(cpuTab);
    GpuTab gpuViewTab = Opm::gpuistl::make_view<GpuV>(gpuBufTab);

    // Evaluation points on the CPU
    Evaluation a(2.3);
    Evaluation b(4.5);

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuA = nullptr;
    Evaluation* gpuB = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuA, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuA, &a, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuB, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuB, &b, sizeof(Evaluation), hipMemcpyHostToDevice));

    gpuComputedResultOnCpu = launchKernelAndRetrieveResult(gpuEvaluateUniformTabulated2DFunction, gpuViewTab, gpuA, gpuB);

    // Free allocated GPU memory
    OPM_GPU_SAFE_CALL(hipFree(gpuA));
    OPM_GPU_SAFE_CALL(hipFree(gpuB));

    // Verify that the CPU and GPU results match within a reasonable tolerance
    const double cpuComputedResult = cpuTab.eval(a, b, true).value();
    BOOST_CHECK(std::fabs(gpuComputedResultOnCpu - cpuComputedResult) < ABS_TOL);
}

// Test case evaluating CO2 pvt properties on CPU and GPU
BOOST_FIXTURE_TEST_CASE(TestUseCO2OnGpu, Fixture) {

    // use the CO2 tables to aquire the viscosity at 290[K] and 2e5[Pa]
    double viscosityReference = Opm::CO2<double, Opm::CO2Tables<double, std::vector<double>>>::gasViscosity(co2Tables, temp, pressure, true).value();

    GpuBufCo2Tables gpuBufCo2Table = Opm::gpuistl::copy_to_gpu<GpuB>(co2Tables);
    GpuViewCO2Tables gpuViewCo2Table = Opm::gpuistl::make_view<GpuV>(gpuBufCo2Table);

    gpuComputedResultOnCpu = launchKernelAndRetrieveResult(gpuCO2GasViscosity, gpuViewCo2Table, gpuTemp, gpuPressure);

    // Verify that the CPU and GPU results match within a reasonable tolerance
    BOOST_CHECK(std::fabs(gpuComputedResultOnCpu - viscosityReference) < ABS_TOL);
}

// Test case evaluating pvt values for SimpleHuDuanH20 on a GPU and CPU
BOOST_FIXTURE_TEST_CASE(TestUseH2OOnGpu, Fixture) {

    // use the CO2 tables to aquire the densityReference at 290[K] and 2e5[Pa]
    double densityReference = HuDuan::liquidDensity(temp, pressure, true).value();

    gpuComputedResultOnCpu = launchKernelAndRetrieveResult(huDuanLiquidDensity, gpuTemp, gpuPressure);

    // Verify that the CPU and GPU results match within a reasonable tolerance
    BOOST_CHECK(std::fabs(gpuComputedResultOnCpu - densityReference) < ABS_TOL);
}

// Test case evaluating pvt values for BrineDynamic on a GPU and CPU
BOOST_FIXTURE_TEST_CASE(TestUseBrineDynamicOnGpu, Fixture) {
    Evaluation salinity(0.1); // [g/Kg]

    // use the CO2 tables to aquire the enthalpyReference at 290[K] and 2e5[Pa]
    double enthalpyReference = BrineDyn::liquidEnthalpy(temp, pressure, salinity).value();

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuSalinity = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuSalinity, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuSalinity, &salinity, sizeof(Evaluation), hipMemcpyHostToDevice));

    gpuComputedResultOnCpu = launchKernelAndRetrieveResult(brineDynamicLiquidEnthalpy, gpuTemp, gpuPressure, gpuSalinity);

    // Verify that the CPU and GPU results match within a reasonable tolerance
    BOOST_CHECK(std::fabs(gpuComputedResultOnCpu - enthalpyReference) < ABS_TOL);
}

// Test case evaluating pvt values for BrineDynamic on a GPU and CPU
BOOST_FIXTURE_TEST_CASE(TestBrine_CO2OnGPU, Fixture) {

    // use the CO2 tables to aquire the gasDiffCoeffReference at 290[K] and 2e5[Pa]
    double gasDiffCoeffReference = CpuBrine_CO2::gasDiffCoeff(co2Tables, temp, pressure, true).value();

    // use the CO2 tables to aquire the viscosity at 290[K] and 2e5[Pa]
    double viscosity = Opm::CO2<double, Opm::CO2Tables<double, std::vector<double>>>::gasViscosity(co2Tables, temp, pressure, true).value();

    GpuBufCo2Tables gpuBufCo2Table = Opm::gpuistl::copy_to_gpu<GpuB>(co2Tables);
    GpuViewCO2Tables gpuViewCo2Table = Opm::gpuistl::make_view<GpuV>(gpuBufCo2Table);

    gpuComputedResultOnCpu = launchKernelAndRetrieveResult(brineCO2GasDiffCoeff, gpuViewCo2Table, gpuTemp, gpuPressure);

    // Verify that the CPU and GPU results match within a reasonable tolerance
    BOOST_CHECK(std::fabs(gpuComputedResultOnCpu - gasDiffCoeffReference) < ABS_TOL);
}

// Test case evaluating pvt values for BrineDynamic on a GPU and CPU
BOOST_FIXTURE_TEST_CASE(TestCo2GasPvt, Fixture) {
    std::vector<double> salinities = {0.2, 0.3, 0.4};

    CpuCo2Pvt cpuCo2Pvt(salinities);
    double internalEnergyReference = cpuCo2Pvt.internalEnergy(1, temp, pressure, Evaluation(0.4), Evaluation(0.0)).value();

    GpuBufCo2Pvt gpuBufCo2Pvt = Opm::gpuistl::copy_to_gpu<GpuB, GpuBufCo2Tables>(cpuCo2Pvt);
    auto brineReferenceDensityCPUCopy = gpuBufCo2Pvt.getBrineReferenceDensity().asStdVector();
    GpuViewCo2Pvt gpuViewCo2Pvt = Opm::gpuistl::make_view<GpuV, GpuViewCO2Tables>(gpuBufCo2Pvt);

    gpuComputedResultOnCpu = launchKernelAndRetrieveResult(co2GasPvtInternalEnergy, gpuViewCo2Pvt, gpuTemp, gpuPressure);

    // Verify that the CPU and GPU results match within a reasonable tolerance
    BOOST_CHECK(compareSignificantDigits(gpuComputedResultOnCpu, internalEnergyReference, 6));
}


BOOST_FIXTURE_TEST_CASE(TestBrineCo2Pvt, Fixture) {
    Evaluation rs(0.3);
    Evaluation saltConcentration(0.1);
    std::vector<double> salinities = {0.2, 0.3, 0.4};

    CpuBrineCo2Pvt cpuBrineCo2Pvt(salinities);
    double internalEnergyReference = cpuBrineCo2Pvt.internalEnergy(1, temp, pressure, rs, saltConcentration).value();

    GpuBufBrineCo2Pvt gpuBufBrineCo2Pvt = Opm::gpuistl::copy_to_gpu<GpuBufCo2Tables, GpuB>(cpuBrineCo2Pvt);
    GpuViewBrineCo2Pvt gpuViewBrineCo2Pvt = Opm::gpuistl::make_view<GpuV, GpuViewCO2Tables>(gpuBufBrineCo2Pvt);

    // Allocate memory for the result on the GPU
    double* resultOnGpu = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&resultOnGpu, sizeof(double)));

    // Allocate GPU memory for the Evaluation inputs
    Evaluation* gpuRs = nullptr;
    Evaluation* gpuSaltConcentration = nullptr;
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuRs, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuRs, &rs, sizeof(Evaluation), hipMemcpyHostToDevice));
    OPM_GPU_SAFE_CALL(hipMalloc(&gpuSaltConcentration, sizeof(Evaluation)));
    OPM_GPU_SAFE_CALL(hipMemcpy(gpuSaltConcentration, &saltConcentration, sizeof(Evaluation), hipMemcpyHostToDevice));

    gpuComputedResultOnCpu = launchKernelAndRetrieveResult(brineCo2PvtInternalEnergy, gpuViewBrineCo2Pvt, gpuTemp, gpuPressure, gpuRs, gpuSaltConcentration);

    BOOST_CHECK(compareSignificantDigits(gpuComputedResultOnCpu, internalEnergyReference, 6));
}
