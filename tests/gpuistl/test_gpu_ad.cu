#include "hip/hip_runtime.h"
/*
  Copyright 2024 SINTEF AS
  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestGpuAD

#include <boost/test/unit_test.hpp>
#include <opm/material/densead/Evaluation.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>
#include <hip/hip_runtime.h>

namespace{
__global__ void instantiate_ad_object(Opm::DenseAd::Evaluation<float, 3>* adObj, double value){
    *adObj = Opm::DenseAd::Evaluation<float, 3>(value, 0);
}

//TODO add more comprenehsive AD tests

} // END EMPTY NAMESPACE


BOOST_AUTO_TEST_CASE(TestInstantiateADObject)
{
    using Evaluation = Opm::DenseAd::Evaluation<float, 3>;
    double testValue = 123.456;
    Evaluation cpuMadeAd = Evaluation(testValue, 0);

    Evaluation gpuMadeAd; // allocate space for one more AD object on the CPU
    Evaluation *d_ad;

    // allocate space on GPU, run kernel, and move results back to the CPU
    OPM_GPU_SAFE_CALL(hipMalloc(&d_ad, sizeof(Evaluation)));
    instantiate_ad_object<<<1,1>>>(d_ad, testValue);
    OPM_GPU_SAFE_CALL(hipDeviceSynchronize());
    OPM_GPU_SAFE_CALL(hipMemcpy(&gpuMadeAd, d_ad, sizeof(Evaluation), hipMemcpyDeviceToHost));
    OPM_GPU_SAFE_CALL(hipFree(d_ad));

    // Check that the object made in a GPU kernel is equivalent to that made on the CPU
    BOOST_CHECK(cpuMadeAd == gpuMadeAd);
}
