#include "hip/hip_runtime.h"
/*
  Copyright 2025 EQUINOR ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestMiniMatrix

#include <boost/test/unit_test.hpp>
#include <hip/hip_runtime.h>
#include <opm/simulators/linalg/gpuistl/MiniMatrix.hpp>
#include <utility> // for std::ignore

using MatType = Opm::gpuistl::MiniMatrix<double, 3>;

template<typename MatType>
__global__ void doNothingKernel(MatType m)
{
    auto idx = threadIdx.x;
    return;
}

BOOST_AUTO_TEST_CASE(TestPassingMatrixToKernel)
{
    MatType m;
    doNothingKernel<<<1, 1>>>(m);
    std::ignore = hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    BOOST_CHECK(err == hipSuccess);
}

template<typename MatType>
__global__ void MiniMatrixOperationsInKernel(MatType m1, MatType m2)
{
    {
      auto tmp = m1 * m2;
    }
    {
      auto tmp = m1 + m2;
    }
    {
      auto tmp = m1 - m2;
    }
    m1 += m2;
    m1 -= m2;

    // Check equality and iterator usage
    for (auto it1 = m1.begin(), it2 = m2.begin(); it1 != m1.end() && it2 != m2.end(); ++it1, ++it2)
    {
        assert(*it1 == *it2);
    }

    return;
}

BOOST_AUTO_TEST_CASE(TestMiniMatrixOperationsInKernel)
{
    MatType m1 = 1.0;
    MatType m2 = 1.0;
    MiniMatrixOperationsInKernel<<<1, 1>>>(m1, m2);
    std::ignore = hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    BOOST_CHECK(err == hipSuccess);
}
