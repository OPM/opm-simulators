#include "hip/hip_runtime.h"
/*
  Copyright 2025 EQUINOR ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#define BOOST_TEST_MODULE TestMiniMatrix

#include <boost/test/unit_test.hpp>
#include <hip/hip_runtime.h>
#include <opm/simulators/linalg/gpuistl/MiniMatrix.hpp>
#include <utility> // for std::ignore

using MatType = Opm::gpuistl::MiniMatrix<double, 3>;

__global__ void doNothingKernel(MatType m)
{
    auto idx = threadIdx.x;
    return;
}

BOOST_AUTO_TEST_CASE(TestPassingMatrixToKernel)
{
    MatType m;
    doNothingKernel<<<1, 1>>>(m);
    std::ignore = hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    BOOST_CHECK(err == hipSuccess);
}

__global__ void MiniMatrixOperationsInKernel(MatType m1, MatType m2)
{
    {
      auto tmp = m1 * m2;
    }
    {
      auto tmp = m1 + m2;
    }
    {
      auto tmp = m1 - m2;
    }
    m1 += m2;
    m1 -= m2;

    // Check equality and iterator usage
    for (auto it1 = m1.begin(), it2 = m2.begin(); it1 != m1.end() && it2 != m2.end(); ++it1, ++it2)
    {
        assert(*it1 == *it2);
    }

    return;
}

BOOST_AUTO_TEST_CASE(TestMiniMatrixOperationsInKernel)
{
    MatType m1 = 1.0;
    MatType m2 = 1.0;
    MiniMatrixOperationsInKernel<<<1, 1>>>(m1, m2);
    std::ignore = hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    BOOST_CHECK(err == hipSuccess);
}

__global__ void WriteToMatrixInKernel(MatType* m)
{
    (*m) = MatType(1.0);
    (*m)[1][1] = 3.14;
    return;
}

BOOST_AUTO_TEST_CASE(TestWritingToMatrixInKernel)
{
    MatType* d_m = nullptr;
    std::ignore = hipMalloc(&d_m, sizeof(MatType));

    WriteToMatrixInKernel<<<1, 1>>>(d_m);
    std::ignore = hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    BOOST_CHECK(err == hipSuccess);

    MatType h_m;
    std::ignore = hipMemcpy(&h_m, d_m, sizeof(MatType), hipMemcpyDeviceToHost);
    std::ignore = hipFree(d_m);

    BOOST_CHECK(h_m[1][1] == 3.14);
}
