#include "hip/hip_runtime.h"
/*
  Copyright 2025 Equinor ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>

#include <opm/simulators/linalg/gpuistl/detail/cpr_amg_operations.hpp>
#include <opm/simulators/linalg/gpuistl/detail/deviceBlockOperations.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpuThreadUtils.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpu_safe_call.hpp>

#include <hip/hip_runtime.h>

namespace Opm::gpuistl::detail
{

namespace
{
    // Kernel for calculating quasi-IMPES weights
    template <typename T, bool transpose, int blockSize>
    __global__ void quasiImpesWeightsKernel(const T* matrix,
                                            T* weights,
                                            const int* diagonalIndices,
                                            const int numberOfRows,
                                            const int pressureVarIndex)
    {
        const auto row = blockDim.x * blockIdx.x + threadIdx.x;

        if (row < numberOfRows) {
            const int diagIdx = diagonalIndices[row];
            const int blockOffset = diagIdx * blockSize * blockSize;
            const T* block = matrix + blockOffset;

            // Set up RHS with 1.0 at pressure index
            T rhs[blockSize] = {0};
            rhs[pressureVarIndex] = 1.0;

            // Storage for solution
            T bweights[blockSize] = {0};

            // Solve the system
            if constexpr (transpose) {
                // Solve using original matrix
                solveBlock<T, blockSize>(block, rhs, bweights);
            } else {
                // Create transposed block for solving
                T transposed[blockSize * blockSize];
                transposeBlock<T, blockSize>(block, transposed);
                solveBlock<T, blockSize>(transposed, rhs, bweights);
            }

            // Find maximum absolute value for normalization
            T invMaxAbs = abs(bweights[0]);
            for (int j = 1; j < blockSize; ++j) {
                invMaxAbs = max(invMaxAbs, abs(bweights[j]));
            }
            invMaxAbs = T(1.0) / invMaxAbs;

            // Normalize and store weights
            for (int j = 0; j < blockSize; ++j) {
                weights[row * blockSize + j] = bweights[j] * invMaxAbs;
            }
        }
    }

    // Kernel to calculate matrix entries for the coarse level - processes each row in parallel
    template <typename T, bool transpose>
    __global__ void calculateCoarseEntriesKernel(const T* fineNonZeroValues,
                                                 T* coarseNonZeroValues,
                                                 const T* weights,
                                                 const int* rowIndices,
                                                 const int* colIndices,
                                                 const int numberOfRows,
                                                 const int blockSize,
                                                 const int pressureVarIndex)
    {
        // Each thread processes one row of the matrix
        const auto row = blockDim.x * blockIdx.x + threadIdx.x;

        if (row < numberOfRows) {
            // Get start and end indices for this row
            const int start = rowIndices[row];
            const int end = rowIndices[row + 1];

            // Process all non-zeros in this row
            for (int i = start; i < end; i++) {
                const int col = colIndices[i];
                const int blockOffset = i * blockSize * blockSize;
                T matrixEl = 0.0;

                if constexpr (transpose) {
                    // Use column weight
                    const T* bw = weights + col * blockSize;
                    for (int j = 0; j < blockSize; ++j) {
                        matrixEl += fineNonZeroValues[blockOffset + pressureVarIndex * blockSize + j] * bw[j];
                    }
                } else {
                    // Use row weight
                    const T* bw = weights + row * blockSize;
                    for (int j = 0; j < blockSize; ++j) {
                        matrixEl += fineNonZeroValues[blockOffset + j * blockSize + pressureVarIndex] * bw[j];
                    }
                }

                coarseNonZeroValues[i] = matrixEl;
            }
        }
    }

    // Kernel to restrict a fine vector to a coarse vector
    template <typename T, bool transpose>
    __global__ void restrictVectorKernel(const T* fine,
                                         T* coarse,
                                         const T* weights,
                                         const int numberOfBlocks,
                                         const int blockSize,
                                         const int pressureVarIndex)
    {
        const auto blockIndex = blockDim.x * blockIdx.x + threadIdx.x;

        if (blockIndex < numberOfBlocks) {
            T rhsEl = 0.0;

            if constexpr (transpose) {
                // Just extract the pressure component
                rhsEl = fine[blockIndex * blockSize + pressureVarIndex];
            } else {
                // Weighted sum of components
                const T* bw = weights + blockIndex * blockSize;
                for (int i = 0; i < blockSize; ++i) {
                    rhsEl += fine[blockIndex * blockSize + i] * bw[i];
                }
            }

            coarse[blockIndex] = rhsEl;
        }
    }

    // Kernel to prolongate a coarse vector to a fine vector
    template <typename T, bool transpose>
    __global__ void prolongateVectorKernel(const T* coarse,
                                           T* fine,
                                           const T* weights,
                                           const int numberOfBlocks,
                                           const int blockSize,
                                           const int pressureVarIndex)
    {
        const auto blockIndex = blockDim.x * blockIdx.x + threadIdx.x;

        if (blockIndex < numberOfBlocks) {
            if constexpr (transpose) {
                // Distribute the coarse value using weights
                const T* bw = weights + blockIndex * blockSize;
                for (int i = 0; i < blockSize; ++i) {
                    fine[blockIndex * blockSize + i] = coarse[blockIndex] * bw[i];
                }
            } else {
                // Only update the pressure component
                fine[blockIndex * blockSize + pressureVarIndex] = coarse[blockIndex];
            }
        }
    }

} // anonymous namespace

template <typename T, bool transpose, int blocksize>
void
dispatchQuasiImpesWeights(const GpuSparseMatrixWrapper<T>& matrix,
                          std::size_t pressureVarIndex,
                          GpuVector<T>& weights,
                          const GpuVector<int>& diagonalIndices,
                          int numberOfRows)
{
    if (matrix.blockSize() != blocksize) {
        if constexpr (blocksize > 1) {
            dispatchQuasiImpesWeights<T, transpose, blocksize - 1>(
                matrix, pressureVarIndex, weights, diagonalIndices, numberOfRows);
        } else {
            throw std::runtime_error("Unsupported block size for getQuasiImpesWeights: " + 
                                   std::to_string(matrix.blockSize()) + ". Only block sizes 1-3 are supported.");
        }
    } else {
        // Launch kernel with the correct block size
        int threadBlockSize = getCudaRecomendedThreadBlockSize(quasiImpesWeightsKernel<T, transpose, blocksize>);
        int nThreadBlocks = getNumberOfBlocks(numberOfRows, threadBlockSize);
        quasiImpesWeightsKernel<T, transpose, blocksize><<<nThreadBlocks, threadBlockSize>>>(
            matrix.getNonZeroValues().data(),
            weights.data(),
            diagonalIndices.data(),
            numberOfRows,
            pressureVarIndex);
    }
}

// Implementation of getQuasiImpesWeights for GPU
template <typename T, bool transpose>
void
getQuasiImpesWeights(const GpuSparseMatrixWrapper<T>& matrix,
                     std::size_t pressureVarIndex,
                     GpuVector<T>& weights,
                     const GpuVector<int>& diagonalIndices)
{
    const int blockSize = matrix.blockSize();
    const int numberOfRows = matrix.N();

    // Ensure weights vector has the right size
    if (weights.dim() != numberOfRows * blockSize) {
        throw std::runtime_error("Weights vector has incorrect size");
    }

    // Dispatch based on block size, max block size is 3
    dispatchQuasiImpesWeights<T, transpose, 3>(matrix, pressureVarIndex, weights, diagonalIndices, numberOfRows);
}

template <typename T, bool transpose>
void
calculateCoarseEntries(const GpuSparseMatrixWrapper<T>& fineMatrix,
                       GpuSparseMatrixWrapper<T>& coarseMatrix,
                       const GpuVector<T>& weights,
                       std::size_t pressureVarIndex)
{
    const int blockSize = fineMatrix.blockSize();
    const int numberOfRows = fineMatrix.N();

    int threadBlockSize = getCudaRecomendedThreadBlockSize(calculateCoarseEntriesKernel<T, transpose>);
    int nThreadBlocks = getNumberOfBlocks(numberOfRows, threadBlockSize);

    calculateCoarseEntriesKernel<T, transpose>
        <<<nThreadBlocks, threadBlockSize>>>(fineMatrix.getNonZeroValues().data(),
                                             coarseMatrix.getNonZeroValues().data(),
                                             weights.data(),
                                             fineMatrix.getRowIndices().data(),
                                             fineMatrix.getColumnIndices().data(),
                                             fineMatrix.N(),
                                             fineMatrix.blockSize(),
                                             pressureVarIndex);
}

template <typename T, bool transpose>
void
restrictVector(const GpuVector<T>& fine,
               GpuVector<T>& coarse,
               const GpuVector<T>& weights,
               std::size_t pressureVarIndex)
{
    const int blockSize = fine.dim() / coarse.dim();
    const int numberOfBlocks = coarse.dim();

    int threadBlockSize = getCudaRecomendedThreadBlockSize(restrictVectorKernel<T, transpose>);
    int nThreadBlocks = getNumberOfBlocks(numberOfBlocks, threadBlockSize);

    restrictVectorKernel<T, transpose><<<nThreadBlocks, threadBlockSize>>>(
        fine.data(), coarse.data(), weights.data(), numberOfBlocks, blockSize, pressureVarIndex);
}

template <typename T, bool transpose>
void
prolongateVector(const GpuVector<T>& coarse,
                 GpuVector<T>& fine,
                 const GpuVector<T>& weights,
                 std::size_t pressureVarIndex)
{
    const int blockSize = fine.dim() / coarse.dim();
    const int numberOfBlocks = coarse.dim();

    int threadBlockSize = getCudaRecomendedThreadBlockSize(prolongateVectorKernel<T, transpose>);
    int nThreadBlocks = getNumberOfBlocks(numberOfBlocks, threadBlockSize);

    prolongateVectorKernel<T, transpose><<<nThreadBlocks, threadBlockSize>>>(
        coarse.data(), fine.data(), weights.data(), numberOfBlocks, blockSize, pressureVarIndex);
}

#define INSTANTIATE_CPR_AMG_FUNCTIONS(ScalarType, TransposeMode)                                                       \
    template void getQuasiImpesWeights<ScalarType, TransposeMode>(const GpuSparseMatrixWrapper<ScalarType>& matrix,           \
                                                                  std::size_t pressureVarIndex,                        \
                                                                  GpuVector<ScalarType>& weights,                      \
                                                                  const GpuVector<int>& diagonalIndices);              \
    template void calculateCoarseEntries<ScalarType, TransposeMode>(const GpuSparseMatrixWrapper<ScalarType>& fineMatrix,     \
                                                                    GpuSparseMatrixWrapper<ScalarType>& coarseMatrix,         \
                                                                    const GpuVector<ScalarType>& weights,              \
                                                                    std::size_t pressureVarIndex);                     \
    template void restrictVector<ScalarType, TransposeMode>(const GpuVector<ScalarType>& fine,                         \
                                                            GpuVector<ScalarType>& coarse,                             \
                                                            const GpuVector<ScalarType>& weights,                      \
                                                            std::size_t pressureVarIndex);                             \
    template void prolongateVector<ScalarType, TransposeMode>(const GpuVector<ScalarType>& coarse,                     \
                                                              GpuVector<ScalarType>& fine,                             \
                                                              const GpuVector<ScalarType>& weights,                    \
                                                              std::size_t pressureVarIndex);

INSTANTIATE_CPR_AMG_FUNCTIONS(double, false)
INSTANTIATE_CPR_AMG_FUNCTIONS(double, true)
INSTANTIATE_CPR_AMG_FUNCTIONS(float, false)
INSTANTIATE_CPR_AMG_FUNCTIONS(float, true)

#undef INSTANTIATE_CPR_AMG_FUNCTIONS

} // namespace Opm::gpuistl::detail
