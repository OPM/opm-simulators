#include "hip/hip_runtime.h"
/*
  Copyright 2024 SINTEF AS

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>
#include <opm/common/ErrorMacros.hpp>
#include <opm/simulators/linalg/gpuistl/detail/deviceBlockOperations.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpuThreadUtils.hpp>
#include <opm/simulators/linalg/gpuistl/detail/preconditionerKernels/JacKernels.hpp>
#include <stdexcept>

namespace Opm::gpuistl::detail::JAC
{
namespace
{
    template <class T, int blocksize>
    __global__ void
    cuInvertDiagonalAndFlatten(T* matNonZeroValues, int* rowIndices, int* colIndices, size_t numberOfRows, T* vec)
    {
        const auto row = blockDim.x * blockIdx.x + threadIdx.x;

        if (row < numberOfRows) {
            size_t nnzIdx = rowIndices[row];
            size_t nnzIdxLim = rowIndices[row + 1];

            // this loop will cause some extra checks that we are within the limit in the case of the diagonal having a
            // zero element
            while (colIndices[nnzIdx] != row && nnzIdx <= nnzIdxLim) {
                ++nnzIdx;
            }

            // diagBlock points to the start of where the diagonal block is stored
            T* diagBlock = &matNonZeroValues[blocksize * blocksize * nnzIdx];
            // vecBlock points to the start of the block element in the vector where the inverse of the diagonal block
            // element should be stored
            T* vecBlock = &vec[blocksize * blocksize * row];

            invBlockOutOfPlace<T, blocksize>(diagBlock, vecBlock);
        }
    }
} // namespace

template <class T, int blocksize>
void
invertDiagonalAndFlatten(T* mat, int* rowIndices, int* colIndices, size_t numberOfRows, T* vec)
{
    if (blocksize <= 3) {
        int threadBlockSize
            = ::Opm::gpuistl::detail::getCudaRecomendedThreadBlockSize(cuInvertDiagonalAndFlatten<T, blocksize>);
        int nThreadBlocks = ::Opm::gpuistl::detail::getNumberOfBlocks(numberOfRows, threadBlockSize);
        cuInvertDiagonalAndFlatten<T, blocksize>
            <<<nThreadBlocks, threadBlockSize>>>(mat, rowIndices, colIndices, numberOfRows, vec);
    } else {
        OPM_THROW(std::invalid_argument, "Inverting diagonal is not implemented for blocksizes > 3");
    }
}

#define INSTANTIATE_KERNEL_WRAPPERS(T, blocksize)                                                                      \
    template void invertDiagonalAndFlatten<T, blocksize>(T*, int*, int*, size_t, T*);

INSTANTIATE_KERNEL_WRAPPERS(float, 1);
INSTANTIATE_KERNEL_WRAPPERS(float, 2);
INSTANTIATE_KERNEL_WRAPPERS(float, 3);
INSTANTIATE_KERNEL_WRAPPERS(float, 4);
INSTANTIATE_KERNEL_WRAPPERS(float, 5);
INSTANTIATE_KERNEL_WRAPPERS(float, 6);
INSTANTIATE_KERNEL_WRAPPERS(double, 1);
INSTANTIATE_KERNEL_WRAPPERS(double, 2);
INSTANTIATE_KERNEL_WRAPPERS(double, 3);
INSTANTIATE_KERNEL_WRAPPERS(double, 4);
INSTANTIATE_KERNEL_WRAPPERS(double, 5);
INSTANTIATE_KERNEL_WRAPPERS(double, 6);

} // namespace Opm::gpuistl::detail::JAC
