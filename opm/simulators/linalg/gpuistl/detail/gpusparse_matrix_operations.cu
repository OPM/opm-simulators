#include "hip/hip_runtime.h"
/*
  Copyright 2022-2023 SINTEF AS

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <config.h>
#include <opm/common/ErrorMacros.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpusparse_matrix_operations.hpp>
#include <opm/simulators/linalg/gpuistl/detail/deviceBlockOperations.hpp>
#include <opm/simulators/linalg/gpuistl/detail/gpuThreadUtils.hpp>
#include <stdexcept>

namespace Opm::gpuistl::detail
{
namespace
{
    template <class T, int blocksize>
    __global__ void cuMoveDataToReordered(
        T* srcMatrix, int* srcRowIndices, T* dstMatrix, int* dstRowIndices, int* indexConversion, size_t numberOfRows)
    {
        const auto srcRow = blockDim.x * blockIdx.x + threadIdx.x;
        if (srcRow < numberOfRows) {

            const auto dstRow = indexConversion[srcRow];

            for (int srcBlock = srcRowIndices[srcRow], dstBlock = dstRowIndices[dstRow];
                 srcBlock < srcRowIndices[srcRow + 1];
                 ++srcBlock, ++dstBlock) {
                for (int i = 0; i < blocksize; ++i) {
                    for (int j = 0; j < blocksize; ++j) {
                        dstMatrix[dstBlock * blocksize * blocksize + i * blocksize + j]
                            = srcMatrix[srcBlock * blocksize * blocksize + i * blocksize + j];
                    }
                }
            }
        }
    }

    template <class T, int blocksize>
    __global__ void cuMoveDataToReorderedSplit(T* srcMatrix,
                                               int* srcRowIndices,
                                               int* srcColumnIndices,
                                               T* dstLowerMatrix,
                                               int* dstLowerRowIndices,
                                               T* dstUpperMatrix,
                                               int* dstUpperRowIndices,
                                               T* dstDiag,
                                               int* naturalToReordered,
                                               size_t numberOfRows)
    {
        const auto srcRow = blockDim.x * blockIdx.x + threadIdx.x;
        if (srcRow < numberOfRows) {

            const auto dstRow = naturalToReordered[srcRow];
            const auto rowStart = srcRowIndices[srcRow];
            const auto rowEnd = srcRowIndices[srcRow + 1];

            auto lowerBlock = dstLowerRowIndices[dstRow];
            auto upperBlock = dstUpperRowIndices[dstRow];

            for (int srcBlock = rowStart; srcBlock < rowEnd; srcBlock++) {
                int dstBlock;
                T* dstBuffer;

                if (srcColumnIndices[srcBlock] < srcRow) { // we are writing a value to the lower triangular matrix
                    dstBlock = lowerBlock;
                    ++lowerBlock;
                    dstBuffer = dstLowerMatrix;
                } else if (srcColumnIndices[srcBlock]
                           > srcRow) { // we are writing a value to the upper triangular matrix
                    dstBlock = upperBlock;
                    ++upperBlock;
                    dstBuffer = dstUpperMatrix;
                } else { // we are writing a value to the diagonal
                    dstBlock = dstRow;
                    dstBuffer = dstDiag;
                }
                for (int i = 0; i < blocksize; ++i) {
                    for (int j = 0; j < blocksize; ++j) {
                        dstBuffer[dstBlock * blocksize * blocksize + i * blocksize + j]
                            = srcMatrix[srcBlock * blocksize * blocksize + i * blocksize + j];
                    }
                }
            }
        }
    }
} // namespace

template <class T, int blocksize>
void
copyMatDataToReordered(T* srcMatrix,
                       int* srcRowIndices,
                       T* dstMatrix,
                       int* dstRowIndices,
                       int* naturalToReordered,
                       size_t numberOfRows,
                       int thrBlockSize)
{
    int threadBlockSize
        = ::Opm::gpuistl::detail::getCudaRecomendedThreadBlockSize(cuMoveDataToReordered<T, blocksize>, thrBlockSize);
    int nThreadBlocks = ::Opm::gpuistl::detail::getNumberOfBlocks(numberOfRows, threadBlockSize);
    cuMoveDataToReordered<T, blocksize><<<nThreadBlocks, threadBlockSize>>>(
        srcMatrix, srcRowIndices, dstMatrix, dstRowIndices, naturalToReordered, numberOfRows);
}

template <class T, int blocksize>
void
copyMatDataToReorderedSplit(T* srcMatrix,
                            int* srcRowIndices,
                            int* srcColumnIndices,
                            T* dstLowerMatrix,
                            int* dstLowerRowIndices,
                            T* dstUpperMatrix,
                            int* dstUpperRowIndices,
                            T* dstDiag,
                            int* naturalToReordered,
                            size_t numberOfRows,
                            int thrBlockSize)
{
    int threadBlockSize = ::Opm::gpuistl::detail::getCudaRecomendedThreadBlockSize(
        cuMoveDataToReorderedSplit<T, blocksize>, thrBlockSize);
    int nThreadBlocks = ::Opm::gpuistl::detail::getNumberOfBlocks(numberOfRows, threadBlockSize);
    cuMoveDataToReorderedSplit<T, blocksize><<<nThreadBlocks, threadBlockSize>>>(srcMatrix,
                                                                                 srcRowIndices,
                                                                                 srcColumnIndices,
                                                                                 dstLowerMatrix,
                                                                                 dstLowerRowIndices,
                                                                                 dstUpperMatrix,
                                                                                 dstUpperRowIndices,
                                                                                 dstDiag,
                                                                                 naturalToReordered,
                                                                                 numberOfRows);
}

#define INSTANTIATE_KERNEL_WRAPPERS(T, blocksize)                                                                      \
    template void copyMatDataToReordered<T, blocksize>(T*, int*, T*, int*, int*, size_t, int);                         \
    template void copyMatDataToReorderedSplit<T, blocksize>(T*, int*, int*, T*, int*, T*, int*, T*, int*, size_t, int);

INSTANTIATE_KERNEL_WRAPPERS(float, 1);
INSTANTIATE_KERNEL_WRAPPERS(float, 2);
INSTANTIATE_KERNEL_WRAPPERS(float, 3);
INSTANTIATE_KERNEL_WRAPPERS(float, 4);
INSTANTIATE_KERNEL_WRAPPERS(float, 5);
INSTANTIATE_KERNEL_WRAPPERS(float, 6);
INSTANTIATE_KERNEL_WRAPPERS(double, 1);
INSTANTIATE_KERNEL_WRAPPERS(double, 2);
INSTANTIATE_KERNEL_WRAPPERS(double, 3);
INSTANTIATE_KERNEL_WRAPPERS(double, 4);
INSTANTIATE_KERNEL_WRAPPERS(double, 5);
INSTANTIATE_KERNEL_WRAPPERS(double, 6);
} // namespace Opm::gpuistl::detail
