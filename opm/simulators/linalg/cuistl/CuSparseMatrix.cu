#include <hip/hip_runtime.h>
#include <opm/simulators/linalg/cuistl/CuSparseMatrix.hpp>
#include <opm/simulators/linalg/cuistl/cusparse_safe_call.hpp>

namespace Opm::cuistl
{

template <class T>
CuSparseMatrix<T>::CuSparseMatrix(const T* nonZeroElements,
                                  const int* rowIndices,
                                  const int* columnIndices,
                                  int numberOfNonzeroElements,
                                  int blockSize,
                                  int numberOfRows)
    : nonZeroElements(nonZeroElements, numberOfNonzeroElements)
    , rowIndices(rowIndices, numberOfRows + 1)
    , columnIndices(columnIndices, numberOfNonzeroElements / blockSize)
    , numberOfNonzeroElements(numberOfNonzeroElements)
    , numberOfRows(numberOfRows)
{
    OPM_CUSPARSE_SAFE_CALL(hipsparseCreateMatDescr(&matrixDescription));
    OPM_CUSPARSE_SAFE_CALL(hipsparseSetMatIndexBase(matrixDescription, baseType));

    // TODO: Are there better options for our matrices? Probably not.
    OPM_CUSPARSE_SAFE_CALL(hipsparseSetMatType(matrixDescription, HIPSPARSE_MATRIX_TYPE_GENERAL));
}

template <class T>
CuSparseMatrix<T>::~CuSparseMatrix()
{
    OPM_CUSPARSE_SAFE_CALL(hipsparseDestroyMatDescr(matrixDescription));
}

template <typename T>
void
CuSparseMatrix<T>::setUpperTriangular()
{
    OPM_CUSPARSE_SAFE_CALL(hipsparseSetMatFillMode(matrixDescription, HIPSPARSE_FILL_MODE_UPPER));
}

template <typename T>
void
CuSparseMatrix<T>::setLowerTriangular()
{
    OPM_CUSPARSE_SAFE_CALL(hipsparseSetMatFillMode(matrixDescription, HIPSPARSE_FILL_MODE_LOWER));
}

template <typename T>
void
CuSparseMatrix<T>::setUnitDiagonal()
{
    OPM_CUSPARSE_SAFE_CALL(hipsparseSetMatDiagType(matrixDescription, HIPSPARSE_DIAG_TYPE_UNIT));
}

template <typename T>
void
CuSparseMatrix<T>::setNonUnitDiagonal()
{
    OPM_CUSPARSE_SAFE_CALL(hipsparseSetMatDiagType(matrixDescription, HIPSPARSE_DIAG_TYPE_NON_UNIT));
}
} // namespace Opm::cuistl
