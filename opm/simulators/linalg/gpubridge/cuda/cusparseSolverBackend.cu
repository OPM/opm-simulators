#include "hip/hip_runtime.h"
/*
  Copyright 2019 Equinor ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>

#include <hip/hip_runtime.h>
#include <sstream>

#include <opm/common/OpmLog/OpmLog.hpp>
#include <dune/common/timer.hh>

#include <opm/simulators/linalg/gpubridge/cuda/cusparseSolverBackend.hpp>
#include <opm/simulators/linalg/gpubridge/cuda/cuWellContributions.hpp>
#include <opm/simulators/linalg/gpubridge/GpuResult.hpp>
#include <opm/simulators/linalg/gpubridge/cuda/cuda_header.hpp>

#include "hipblas.h"
#include "hipsparse.h"
// For more information about cusparse, check https://docs.nvidia.com/cuda/cusparse/index.html

// iff true, the nonzeroes of the matrix are copied row-by-row into a contiguous, pinned memory array, then a single GPU memcpy is done
// otherwise, the nonzeroes of the matrix are assumed to be in a contiguous array, and a single GPU memcpy is enough
#define COPY_ROW_BY_ROW 0

#include <thread>
#include <type_traits>

extern std::shared_ptr<std::thread> copyThread;

#if HAVE_OPENMP
#include <omp.h>
#endif // HAVE_OPENMP

namespace Opm::Accelerator {

using Dune::Timer;

const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
const hipsparseOperation_t operation  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
const hipsparseDirection_t order = HIPSPARSE_DIRECTION_ROW;

template<class Scalar, unsigned int block_size>
cusparseSolverBackend<Scalar, block_size>::
cusparseSolverBackend(int verbosity_, int maxit_,
                      Scalar tolerance_, unsigned int deviceID_)
    : Base(verbosity_, maxit_, tolerance_, deviceID_)
{
    // initialize CUDA device, stream and libraries
    hipSetDevice(deviceID);
    cudaCheckLastError("Could not get device");
    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceID);
    cudaCheckLastError("Could not get device properties");
    std::ostringstream out;
    out << "Name GPU: " << props.name << ", Compute Capability: "
        << props.major << "." << props.minor;
    OpmLog::info(out.str());

    hipStreamCreate(&stream);
    cudaCheckLastError("Could not create stream");

    hipblasCreate(&cublasHandle);
    cudaCheckLastError("Could not create cublasHandle");
    hipsparseCreate(&cusparseHandle);
    cudaCheckLastError("Could not create cusparseHandle");

    hipblasSetStream(cublasHandle, stream);
    cudaCheckLastError("Could not set stream to cublas");
    hipsparseSetStream(cusparseHandle, stream);
    cudaCheckLastError("Could not set stream to cusparse");
}

template<class Scalar, unsigned int block_size>
cusparseSolverBackend<Scalar,block_size>::~cusparseSolverBackend()
{
    finalize();
}

template<class Scalar, unsigned int block_size>
void cusparseSolverBackend<Scalar,block_size>::
gpu_pbicgstab(WellContributions<Scalar>& wellContribs, GpuResult& res)
{
    Timer t_total, t_prec(false), t_spmv(false), t_well(false), t_rest(false);
    int n = N;
    Scalar rho = 1.0, rhop;
    Scalar alpha, nalpha, beta;
    Scalar omega, nomega, tmp1, tmp2;
    Scalar norm, norm_0;
    Scalar zero = 0.0;
    Scalar one  = 1.0;
    Scalar mone = -1.0;
    float it;

    if (wellContribs.getNumWells() > 0) {
        static_cast<WellContributionsCuda<Scalar>&>(wellContribs).setCudaStream(stream);
    }

    if constexpr (std::is_same_v<Scalar,float>) {
        hipsparseSbsrmv(cusparseHandle, order, operation, Nb, Nb, nnzb, &one,
                       descr_M, d_bVals, d_bRows, d_bCols, block_size, d_x, &zero, d_r);
    } else {
        hipsparseDbsrmv(cusparseHandle, order, operation, Nb, Nb, nnzb, &one,
                       descr_M, d_bVals, d_bRows, d_bCols, block_size, d_x, &zero, d_r);
    }

    if constexpr (std::is_same_v<Scalar,float>) {
        hipblasSscal(cublasHandle, n, &mone, d_r, 1);
        hipblasSaxpy(cublasHandle, n, &one, d_b, 1, d_r, 1);
        hipblasScopy(cublasHandle, n, d_r, 1, d_rw, 1);
        hipblasScopy(cublasHandle, n, d_r, 1, d_p, 1);
        hipblasSnrm2(cublasHandle, n, d_r, 1, &norm_0);
    } else {
        hipblasDscal(cublasHandle, n, &mone, d_r, 1);
        hipblasDaxpy(cublasHandle, n, &one, d_b, 1, d_r, 1);
        hipblasDcopy(cublasHandle, n, d_r, 1, d_rw, 1);
        hipblasDcopy(cublasHandle, n, d_r, 1, d_p, 1);
        hipblasDnrm2(cublasHandle, n, d_r, 1, &norm_0);
    }

    if (verbosity > 1) {
        std::ostringstream out;
        out << std::scientific << "cusparseSolver initial norm: " << norm_0;
        OpmLog::info(out.str());
    }

    for (it = 0.5; it < maxit; it += 0.5) {
        rhop = rho;
        if constexpr (std::is_same_v<Scalar,float>) {
            hipblasSdot(cublasHandle, n, d_rw, 1, d_r, 1, &rho);
        } else {
            hipblasDdot(cublasHandle, n, d_rw, 1, d_r, 1, &rho);
        }

        if (it > 1) {
            beta = (rho / rhop) * (alpha / omega);
            nomega = -omega;
            if constexpr (std::is_same_v<Scalar,float>) {
                hipblasSaxpy(cublasHandle, n, &nomega, d_v, 1, d_p, 1);
                hipblasSscal(cublasHandle, n, &beta, d_p, 1);
                hipblasSaxpy(cublasHandle, n, &one, d_r, 1, d_p, 1);
            } else {
                hipblasDaxpy(cublasHandle, n, &nomega, d_v, 1, d_p, 1);
                hipblasDscal(cublasHandle, n, &beta, d_p, 1);
                hipblasDaxpy(cublasHandle, n, &one, d_r, 1, d_p, 1);
            }
        }

        if constexpr (std::is_same_v<Scalar,float>) {
            // apply ilu0
            hipsparseSbsrsv2_solve(cusparseHandle, order,
                                  operation, Nb, nnzbs_prec, &one,
                                  descr_L, d_mVals, d_mRows, d_mCols, block_size,
                                  info_L, d_p, d_t, policy, d_buffer);
            hipsparseSbsrsv2_solve(cusparseHandle, order,
                                  operation, Nb, nnzbs_prec, &one,
                                  descr_U, d_mVals, d_mRows, d_mCols, block_size,
                                  info_U, d_t, d_pw, policy, d_buffer);
            // spmv
            hipsparseSbsrmv(cusparseHandle, order,
                           operation, Nb, Nb, nnzb,
                           &one, descr_M, d_bVals, d_bRows,
                           d_bCols, block_size, d_pw, &zero, d_v);
        } else {
            // apply ilu0
            hipsparseDbsrsv2_solve(cusparseHandle, order,
                                       operation, Nb, nnzbs_prec, &one,
                                       descr_L, d_mVals, d_mRows, d_mCols, block_size,
                                       info_L, d_p, d_t, policy, d_buffer);
            hipsparseDbsrsv2_solve(cusparseHandle, order,
                                  operation, Nb, nnzbs_prec, &one,
                                  descr_U, d_mVals, d_mRows, d_mCols, block_size,
                                  info_U, d_t, d_pw, policy, d_buffer);
            // spmv
            hipsparseDbsrmv(cusparseHandle, order,
                           operation, Nb, Nb, nnzb,
                           &one, descr_M, d_bVals, d_bRows, d_bCols, block_size,
                           d_pw, &zero, d_v);
        }

        // apply wellContributions
        if (wellContribs.getNumWells() > 0) {
            static_cast<WellContributionsCuda<Scalar>&>(wellContribs).apply(d_pw, d_v);
        }

        if constexpr (std::is_same_v<Scalar,float>) {
            hipblasSdot(cublasHandle, n, d_rw, 1, d_v, 1, &tmp1);
        } else {
            hipblasDdot(cublasHandle, n, d_rw, 1, d_v, 1, &tmp1);
        }

        alpha = rho / tmp1;
        nalpha = -alpha;
        if constexpr (std::is_same_v<Scalar,float>) {
            hipblasSaxpy(cublasHandle, n, &nalpha, d_v, 1, d_r, 1);
            hipblasSaxpy(cublasHandle, n, &alpha, d_pw, 1, d_x, 1);
            hipblasSnrm2(cublasHandle, n, d_r, 1, &norm);
        } else {
            hipblasDaxpy(cublasHandle, n, &nalpha, d_v, 1, d_r, 1);
            hipblasDaxpy(cublasHandle, n, &alpha, d_pw, 1, d_x, 1);
            hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);
        }

        if (norm < tolerance * norm_0) {
            break;
        }

        it += 0.5;

        if constexpr (std::is_same_v<Scalar,float>) {
            // apply ilu0
            hipsparseSbsrsv2_solve(cusparseHandle, order,
                                  operation, Nb, nnzbs_prec, &one,
                                  descr_L, d_mVals, d_mRows, d_mCols, block_size,
                                  info_L, d_r, d_t, policy, d_buffer);

            hipsparseSbsrsv2_solve(cusparseHandle, order,
                                  operation, Nb, nnzbs_prec, &one,
                                  descr_U, d_mVals, d_mRows, d_mCols, block_size,
                                  info_U, d_t, d_s, policy, d_buffer);

            // spmv
            hipsparseSbsrmv(cusparseHandle, order,
                           operation, Nb, Nb, nnzb, &one, descr_M,
                           d_bVals, d_bRows, d_bCols, block_size, d_s, &zero, d_t);
        } else {
            // apply ilu0
            hipsparseDbsrsv2_solve(cusparseHandle, order,
                                       operation, Nb, nnzbs_prec, &one,
                                       descr_L, d_mVals, d_mRows, d_mCols, block_size,
                                       info_L, d_r, d_t, policy, d_buffer);

            hipsparseDbsrsv2_solve(cusparseHandle, order,
                                  operation, Nb, nnzbs_prec, &one,
                                  descr_U, d_mVals, d_mRows, d_mCols, block_size,
                                  info_U, d_t, d_s, policy, d_buffer);

            // spmv
            hipsparseDbsrmv(cusparseHandle, order,
                           operation, Nb, Nb, nnzb, &one, descr_M,
                           d_bVals, d_bRows, d_bCols, block_size, d_s, &zero, d_t);
        }

        // apply wellContributions
        if (wellContribs.getNumWells() > 0) {
            static_cast<WellContributionsCuda<Scalar>&>(wellContribs).apply(d_s, d_t);
        }

        if constexpr (std::is_same_v<Scalar,float>) {
            hipblasSdot(cublasHandle, n, d_t, 1, d_r, 1, &tmp1);
            hipblasSdot(cublasHandle, n, d_t, 1, d_t, 1, &tmp2);
        } else {
            hipblasDdot(cublasHandle, n, d_t, 1, d_r, 1, &tmp1);
            hipblasDdot(cublasHandle, n, d_t, 1, d_t, 1, &tmp2);
        }

        omega = tmp1 / tmp2;
        nomega = -omega;

        if constexpr (std::is_same_v<Scalar,float>) {
            hipblasSaxpy(cublasHandle, n, &omega, d_s, 1, d_x, 1);
            hipblasSaxpy(cublasHandle, n, &nomega, d_t, 1, d_r, 1);
            hipblasSnrm2(cublasHandle, n, d_r, 1, &norm);
        } else {
            hipblasDaxpy(cublasHandle, n, &omega, d_s, 1, d_x, 1);
            hipblasDaxpy(cublasHandle, n, &nomega, d_t, 1, d_r, 1);
            hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);
        }

        if (norm < tolerance * norm_0) {
            break;
        }

        if (verbosity > 1) {
            std::ostringstream out;
            out << "it: " << it << std::scientific << ", norm: " << norm;
            OpmLog::info(out.str());
        }
    }

    res.iterations = std::min(it, (float)maxit);
    res.reduction = norm / norm_0;
    res.conv_rate  = static_cast<double>(pow(res.reduction, 1.0 / it));
    res.elapsed = t_total.stop();
    res.converged = (it != (maxit + 0.5));

    if (verbosity > 0) {
        std::ostringstream out;
        out << "=== converged: " << res.converged << ", conv_rate: "
            << res.conv_rate << ", time: " << res.elapsed
            << ", time per iteration: " << res.elapsed / it << ", iterations: " << it;
        OpmLog::info(out.str());
    }
}

template<class Scalar, unsigned int block_size>
void cusparseSolverBackend<Scalar,block_size>::
initialize(std::shared_ptr<BlockedMatrix<Scalar>> matrix,
           std::shared_ptr<BlockedMatrix<Scalar>> jacMatrix)
{
    this->Nb = matrix->Nb;
    this->N = Nb * block_size;
    this->nnzb = matrix->nnzbs;
    this->nnz = nnzb * block_size * block_size;

    if (jacMatrix) {
        useJacMatrix = true;
        nnzbs_prec = jacMatrix->nnzbs;
    } else {
        nnzbs_prec = nnzb;
    }

    std::ostringstream out;
    out << "Initializing GPU, matrix size: " << Nb
        << " blockrows, nnz: " << nnzb << " blocks\n";
    if (useJacMatrix) {
        out << "Blocks in ILU matrix: " << nnzbs_prec << "\n";
    }
    out << "Maxit: " << maxit << std::scientific
        << ", tolerance: " << tolerance << "\n";
    OpmLog::info(out.str());

    hipMalloc((void**)&d_x, sizeof(Scalar) * N);
    hipMalloc((void**)&d_b, sizeof(Scalar) * N);
    hipMalloc((void**)&d_r, sizeof(Scalar) * N);
    hipMalloc((void**)&d_rw, sizeof(Scalar) * N);
    hipMalloc((void**)&d_p, sizeof(Scalar) * N);
    hipMalloc((void**)&d_pw, sizeof(Scalar) * N);
    hipMalloc((void**)&d_s, sizeof(Scalar) * N);
    hipMalloc((void**)&d_t, sizeof(Scalar) * N);
    hipMalloc((void**)&d_v, sizeof(Scalar) * N);
    hipMalloc((void**)&d_bVals, sizeof(Scalar) * nnz);
    hipMalloc((void**)&d_bCols, sizeof(int) * nnzb);
    hipMalloc((void**)&d_bRows, sizeof(int) * (Nb + 1));
    if (useJacMatrix) {
        hipMalloc((void**)&d_mVals, sizeof(Scalar) * nnzbs_prec * block_size * block_size);
        hipMalloc((void**)&d_mCols, sizeof(int) * nnzbs_prec);
        hipMalloc((void**)&d_mRows, sizeof(int) * (Nb + 1));
    } else {
        hipMalloc((void**)&d_mVals, sizeof(Scalar) * nnz);
        d_mCols = d_bCols;
        d_mRows = d_bRows;
    }
    cudaCheckLastError("Could not allocate enough memory on GPU");

#if COPY_ROW_BY_ROW
    hipHostMalloc((void**)&vals_contiguous, sizeof(Scalar) * nnz);
    cudaCheckLastError("Could not allocate pinned memory");
#endif

    initialized = true;
} // end initialize()

template<class Scalar, unsigned int block_size>
void cusparseSolverBackend<Scalar,block_size>::finalize()
{
    if (initialized) {
        hipFree(d_x);
        hipFree(d_b);
        hipFree(d_r);
        hipFree(d_rw);
        hipFree(d_p);
        hipFree(d_pw);
        hipFree(d_s);
        hipFree(d_t);
        hipFree(d_v);
        hipFree(d_mVals);
        if (useJacMatrix) {
            hipFree(d_mCols);
            hipFree(d_mRows);
        }
        hipFree(d_bVals);
        hipFree(d_bCols);
        hipFree(d_bRows);
        hipFree(d_buffer);
        hipsparseDestroyBsrilu02Info(info_M);
        hipsparseDestroyBsrsv2Info(info_L);
        hipsparseDestroyBsrsv2Info(info_U);
        hipsparseDestroyMatDescr(descr_B);
        hipsparseDestroyMatDescr(descr_M);
        hipsparseDestroyMatDescr(descr_L);
        hipsparseDestroyMatDescr(descr_U);
        hipsparseDestroy(cusparseHandle);
        hipblasDestroy(cublasHandle);
#if COPY_ROW_BY_ROW
        hipHostFree(vals_contiguous);
#endif
        hipStreamDestroy(stream);
    }
} // end finalize()

template<class Scalar, unsigned int block_size>
void cusparseSolverBackend<Scalar,block_size>::
copy_system_to_gpu(std::shared_ptr<BlockedMatrix<Scalar>> matrix,
                   Scalar* b,
                   std::shared_ptr<BlockedMatrix<Scalar>> jacMatrix)
{
    Timer t;

    hipMemcpyAsync(d_bCols, matrix->colIndices, nnzb * sizeof(int),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_bRows, matrix->rowPointers, (Nb + 1) * sizeof(int),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b, N * sizeof(Scalar), hipMemcpyHostToDevice, stream);
    hipMemsetAsync(d_x, 0, N * sizeof(Scalar), stream);

#if COPY_ROW_BY_ROW
    int sum = 0;
    for (int i = 0; i < Nb; ++i) {
        int size_row = matrix->rowPointers[i + 1] - matrix->rowPointers[i];
        memcpy(vals_contiguous + sum, matrix->nnzValues + sum,
               size_row * sizeof(Scalar) * block_size * block_size);
        sum += size_row * block_size * block_size;
    }
    hipMemcpyAsync(d_bVals, vals_contiguous,
                    nnz * sizeof(Scalar), hipMemcpyHostToDevice, stream);
#else
    hipMemcpyAsync(d_bVals, matrix->nnzValues,
                    nnz * sizeof(Scalar), hipMemcpyHostToDevice, stream);

    bool use_multithreading = true;
#if HAVE_OPENMP
    if(omp_get_max_threads() == 1)
        use_multithreading = false;
#endif

    if (useJacMatrix) {
        if(use_multithreading)
            copyThread->join();

        hipMemcpyAsync(d_mVals, jacMatrix->nnzValues,
                        nnzbs_prec * block_size * block_size * sizeof(Scalar),
                        hipMemcpyHostToDevice, stream);
    } else {
        hipMemcpyAsync(d_mVals, d_bVals,
                        nnz  * sizeof(Scalar),
                        hipMemcpyDeviceToDevice, stream);
    }
#endif

    if (useJacMatrix) {
        hipMemcpyAsync(d_mCols, jacMatrix->colIndices, nnzbs_prec * sizeof(int),
                        hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_mRows, jacMatrix->rowPointers, (Nb + 1) * sizeof(int),
                        hipMemcpyHostToDevice, stream);
    }

    if (verbosity >= 3) {
        hipStreamSynchronize(stream);
        
        c_copy += t.stop();
        std::ostringstream out;
        out << "---cusparseSolver::copy_system_to_gpu(): " << t.elapsed() << " s";
        OpmLog::info(out.str());
    }
} // end copy_system_to_gpu()

// don't copy rowpointers and colindices, they stay the same
template<class Scalar, unsigned int block_size>
void cusparseSolverBackend<Scalar,block_size>::
update_system_on_gpu(std::shared_ptr<BlockedMatrix<Scalar>> matrix,
                     Scalar* b,
                     std::shared_ptr<BlockedMatrix<Scalar>> jacMatrix)
{
    Timer t;

    hipMemcpyAsync(d_b, b, N * sizeof(Scalar), hipMemcpyHostToDevice, stream);
    hipMemsetAsync(d_x, 0, sizeof(Scalar) * N, stream);
    
#if COPY_ROW_BY_ROW
    int sum = 0;
    for (int i = 0; i < Nb; ++i) {
        int size_row = matrix->rowPointers[i + 1] - matrix->rowPointers[i];
        memcpy(vals_contiguous + sum, matrix->nnzValues + sum,
               size_row * sizeof(Scalar) * block_size * block_size);
        sum += size_row * block_size * block_size;
    }
    hipMemcpyAsync(d_bVals, vals_contiguous,
                    nnz * sizeof(Scalar), hipMemcpyHostToDevice, stream);
#else
    hipMemcpyAsync(d_bVals, matrix->nnzValues,
                    nnz * sizeof(Scalar), hipMemcpyHostToDevice, stream);

    bool use_multithreading = true;
#if HAVE_OPENMP
    if (omp_get_max_threads() == 1)
        use_multithreading = false;
#endif

    if (useJacMatrix) {
        if (use_multithreading)
            copyThread->join();

        hipMemcpyAsync(d_mVals, jacMatrix->nnzValues, 
                        nnzbs_prec * block_size * block_size * sizeof(Scalar),
                        hipMemcpyHostToDevice, stream);
    } else {
        hipMemcpyAsync(d_mVals, d_bVals, nnz  * sizeof(Scalar),
                        hipMemcpyDeviceToDevice, stream);
    }
#endif

    if (verbosity >= 3) {
        hipStreamSynchronize(stream);

        c_copy += t.stop();
        std::ostringstream out;
        out << "-----cusparseSolver::update_system_on_gpu(): " << t.elapsed() << " s\n";
        out << "---cusparseSolver::cum copy: " << c_copy << " s";
        OpmLog::info(out.str());
    }
} // end update_system_on_gpu()

template<class Scalar, unsigned int block_size>
bool cusparseSolverBackend<Scalar,block_size>::analyse_matrix()
{
    int d_bufferSize_M, d_bufferSize_L, d_bufferSize_U, d_bufferSize;
    Timer t;

    hipsparseCreateMatDescr(&descr_B);
    hipsparseCreateMatDescr(&descr_M);
    hipsparseSetMatType(descr_B, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
    const hipsparseIndexBase_t base_type = HIPSPARSE_INDEX_BASE_ZERO;     // matrices from Flow are base0

    hipsparseSetMatIndexBase(descr_B, base_type);
    hipsparseSetMatIndexBase(descr_M, base_type);

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, base_type);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatIndexBase(descr_U, base_type);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    cudaCheckLastError("Could not initialize matrix descriptions");

    hipsparseCreateBsrilu02Info(&info_M);
    hipsparseCreateBsrsv2Info(&info_L);
    hipsparseCreateBsrsv2Info(&info_U);
    cudaCheckLastError("Could not create analysis info");

    if constexpr (std::is_same_v<Scalar,float>) {
        hipsparseSbsrilu02_bufferSize(cusparseHandle, order, Nb, nnzbs_prec,
                                     descr_M, d_mVals, d_mRows, d_mCols, block_size,
                                     info_M, &d_bufferSize_M);
        hipsparseSbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzbs_prec,
                                   descr_L, d_mVals, d_mRows, d_mCols, block_size,
                                   info_L, &d_bufferSize_L);
        hipsparseSbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzbs_prec,
                                   descr_U, d_mVals, d_mRows, d_mCols, block_size,
                                   info_U, &d_bufferSize_U);
    } else {
        hipsparseDbsrilu02_bufferSize(cusparseHandle, order, Nb, nnzbs_prec,
                                     descr_M, d_mVals, d_mRows, d_mCols, block_size,
                                     info_M, &d_bufferSize_M);
        hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzbs_prec,
                                   descr_L, d_mVals, d_mRows, d_mCols, block_size,
                                   info_L, &d_bufferSize_L);
        hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzbs_prec,
                                   descr_U, d_mVals, d_mRows, d_mCols, block_size,
                                   info_U, &d_bufferSize_U);
    }

    d_bufferSize = std::max(d_bufferSize_M, std::max(d_bufferSize_L, d_bufferSize_U));

    hipMalloc((void**)&d_buffer, d_bufferSize);

    // analysis of ilu LU decomposition
    if constexpr (std::is_same_v<Scalar,float>) {
        hipsparseSbsrilu02_analysis(cusparseHandle, order,
                                   Nb, nnzbs_prec, descr_B, d_mVals, d_mRows, d_mCols,
                                   block_size, info_M, policy, d_buffer);
    } else {
        hipsparseDbsrilu02_analysis(cusparseHandle, order,
                                   Nb, nnzbs_prec, descr_B, d_mVals, d_mRows, d_mCols,
                                   block_size, info_M, policy, d_buffer);
    }

    int structural_zero;
    hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        return false;
    }

    // analysis of ilu apply
    if constexpr (std::is_same_v<Scalar,float>) {
        hipsparseSbsrsv2_analysis(cusparseHandle, order, operation,
                                 Nb, nnzbs_prec, descr_L, d_mVals, d_mRows, d_mCols,
                                 block_size, info_L, policy, d_buffer);
        hipsparseSbsrsv2_analysis(cusparseHandle, order, operation,
                                 Nb, nnzbs_prec, descr_U, d_mVals, d_mRows, d_mCols,
                                 block_size, info_U, policy, d_buffer);
    } else {
        hipsparseDbsrsv2_analysis(cusparseHandle, order, operation,
                                 Nb, nnzbs_prec, descr_L, d_mVals, d_mRows, d_mCols,
                                 block_size, info_L, policy, d_buffer);
        hipsparseDbsrsv2_analysis(cusparseHandle, order, operation,
                                 Nb, nnzbs_prec, descr_U, d_mVals, d_mRows, d_mCols,
                                 block_size, info_U, policy, d_buffer);
    }
    cudaCheckLastError("Could not analyse level information");

    if (verbosity > 2) {
        hipStreamSynchronize(stream);
        std::ostringstream out;
        out << "cusparseSolver::analyse_matrix(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }

    analysis_done = true;

    return true;
} // end analyse_matrix()

template<class Scalar, unsigned int block_size>
bool cusparseSolverBackend<Scalar,block_size>::create_preconditioner()
{
    Timer t;

    if constexpr (std::is_same_v<Scalar,float>) {
        hipsparseSbsrilu02(cusparseHandle, order,
                          Nb, nnzbs_prec, descr_M, d_mVals, d_mRows, d_mCols,
                          block_size, info_M, policy, d_buffer);
    } else {
        hipsparseDbsrilu02(cusparseHandle, order,
                          Nb, nnzbs_prec, descr_M, d_mVals, d_mRows, d_mCols,
                          block_size, info_M, policy, d_buffer);
    }
    cudaCheckLastError("Could not perform ilu decomposition");

    int structural_zero;
    // hipsparseXbsrilu02_zeroPivot() calls hipDeviceSynchronize()
    hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        return false;
    }

    if (verbosity > 2) {
        hipStreamSynchronize(stream);
        std::ostringstream out;
        out << "cusparseSolver::create_preconditioner(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }
    return true;
} // end create_preconditioner()

template<class Scalar, unsigned int block_size>
void cusparseSolverBackend<Scalar,block_size>::
solve_system(WellContributions<Scalar>& wellContribs, GpuResult& res)
{
    // actually solve
    gpu_pbicgstab(wellContribs, res);
    hipStreamSynchronize(stream);
    cudaCheckLastError("Something went wrong during the GPU solve");
} // end solve_system()

// copy result to host memory
// caller must be sure that x is a valid array
template<class Scalar, unsigned int block_size>
void cusparseSolverBackend<Scalar,block_size>::get_result(Scalar* x)
{
    Timer t;

    hipMemcpyAsync(x, d_x, N * sizeof(Scalar), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    if (verbosity > 2) {
        std::ostringstream out;
        out << "cusparseSolver::get_result(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }
} // end get_result()

template<class Scalar, unsigned int block_size>
SolverStatus cusparseSolverBackend<Scalar,block_size>::
solve_system(std::shared_ptr<BlockedMatrix<Scalar>> matrix,
             Scalar* b,
             std::shared_ptr<BlockedMatrix<Scalar>> jacMatrix,
             WellContributions<Scalar>& wellContribs,
             GpuResult& res)
{
    if (initialized == false) {
        initialize(matrix, jacMatrix);
        copy_system_to_gpu(matrix, b, jacMatrix);
    } else {
        update_system_on_gpu(matrix, b, jacMatrix);
    }
    if (analysis_done == false) {
        if (!analyse_matrix()) {
            return SolverStatus::GPU_SOLVER_ANALYSIS_FAILED;
        }
    }
    if (create_preconditioner()) {
        solve_system(wellContribs, res);
    } else {
        return SolverStatus::GPU_SOLVER_CREATE_PRECONDITIONER_FAILED;
    }
    return SolverStatus::GPU_SOLVER_SUCCESS;
}

#define INSTANTIATE_TYPE(T)                    \
    template class cusparseSolverBackend<T,1>; \
    template class cusparseSolverBackend<T,2>; \
    template class cusparseSolverBackend<T,3>; \
    template class cusparseSolverBackend<T,4>; \
    template class cusparseSolverBackend<T,5>; \
    template class cusparseSolverBackend<T,6>;

INSTANTIATE_TYPE(double)

#if FLOW_INSTANTIATE_FLOAT
INSTANTIATE_TYPE(float)
#endif

} // namespace Opm::Accelerator
