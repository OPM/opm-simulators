#include "hip/hip_runtime.h"
/*
  Copyright 2020 Equinor ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <config.h> // CMake
#include <cstdlib>
#include <cstring>

#include "opm/simulators/linalg/bda/cuWellContributions.hpp"

#include "opm/simulators/linalg/bda/cuda_header.hpp"
#include <hip/hip_runtime.h>

#include <opm/common/OpmLog/OpmLog.hpp>
#include <opm/common/ErrorMacros.hpp>

namespace Opm
{

// apply WellContributions using y -= C^T * (D^-1 * (B * x))
__global__ void apply_well_contributions(
    const double * __restrict__ Cnnzs,
    const double * __restrict__ Dnnzs,
    const double * __restrict__ Bnnzs,
    const int * __restrict__ Ccols,
    const int * __restrict__ Bcols,
    const double * __restrict__ x,
    double * __restrict__ y,
    const int dim,
    const int dim_wells,
    const unsigned int * __restrict__ val_pointers
)
{
    const int idx_b = blockIdx.x;
    const int idx_t = threadIdx.x;
    const unsigned int val_size = val_pointers[idx_b + 1] - val_pointers[idx_b];

    const int vals_per_block = dim * dim_wells;        // 12
    const int num_active_threads = (32 / vals_per_block) * vals_per_block; // 24
    const int num_blocks_per_warp = 32 / vals_per_block; // 2
    const int lane = idx_t % 32;
    const int c = lane % dim;                           // col in block
    const int r = (lane / dim) % dim_wells;             // row in block

    extern __shared__ double smem[];
    double * __restrict__ z1 = smem;
    double * __restrict__ z2 = z1 + dim_wells;

    if (idx_t < dim_wells) {
        z1[idx_t] = 0.0;
    }

    __syncthreads();

    // z1 = B * x
    if (idx_t < num_active_threads) {
        // multiply all blocks with x
        double temp = 0.0;
        int b = idx_t / vals_per_block + val_pointers[idx_b];       // block id, val_size indicates number of blocks
        while (b < val_size + val_pointers[idx_b]) {
            int colIdx = Bcols[b];
            temp += Bnnzs[b * dim * dim_wells + r * dim + c] * x[colIdx * dim + c];
            b += num_blocks_per_warp;
        }

        // merge all blocks into 1 dim*dim_wells block
        // since 3*4 blocks has give 2 parallel blocks, do not use a loop
        // 0x00ffffff contains 24 ones, representing the two blocks that are added
        // block 1:     block 2:
        //  0  1  2     12 13 14
        //  3  4  5     15 16 17
        //  6  7  8     18 19 20
        //  9 10 11     21 22 23
        // thread i will hold the sum of thread i and i + vals_per_block
        temp += __shfl_down_sync(0x00ffffff, temp, dim * dim_wells);

        // merge all (dim) columns of 1 block, results in a single 1*dim_wells vector, which is used to multiply with invD
        if (idx_t < vals_per_block) {
            // should be a loop as well, now only works for dim == 3
            if (c == 0 || c == 2) {temp += __shfl_down_sync(0x00000B6D, temp, 2);} // add col 2 to col 0
            if (c == 0 || c == 1) {temp += __shfl_down_sync(0x000006DB, temp, 1);} // add col 1 to col 0
        }

        // write 1*dim_wells vector to gmem, could be replaced with shfl broadcast to remove z1 altogether
        if (c == 0 && idx_t < vals_per_block) {
            z1[r] = temp;
        }
    }

    __syncthreads();

    // z2 = D^-1 * B * x = D^-1 * z1
    if (idx_t < dim_wells) {
        double temp = 0.0;
        for (int c = 0; c < dim_wells; ++c) {
            temp += Dnnzs[idx_b * dim_wells * dim_wells + idx_t * dim_wells + c] * z1[c];
        }
        z2[idx_t] = temp;
    }

    __syncthreads();

    // y -= C^T * D^-1 * B * x
    // use dim * val_size threads, each block is assigned 'dim' threads
    if (idx_t < dim * val_size) {
        double temp = 0.0;
        int b = idx_t / dim + val_pointers[idx_b];
        int cc = idx_t % dim;
        int colIdx = Ccols[b];
        for (unsigned int c = 0; c < dim_wells; ++c) {
            temp += Cnnzs[b * dim * dim_wells + c * dim + cc] * z2[c];
        }
        y[colIdx * dim + cc] -= temp;
    }

}

WellContributionsCuda::~WellContributionsCuda()
{
    // delete data for StandardWell
    if (num_std_wells > 0) {
        hipFree(d_Cnnzs);
        hipFree(d_Dnnzs);
        hipFree(d_Bnnzs);
        hipFree(d_Ccols);
        hipFree(d_Bcols);
        hipFree(d_val_pointers);
    }

    if (num_ms_wells > 0 && h_x) {
        hipHostFree(h_x);
        hipHostFree(h_y);
        h_x = h_y = nullptr; // Mark as free for constructor
    }
}

void WellContributionsCuda::APIalloc()
{
    hipMalloc((void**)&d_Cnnzs, sizeof(double) * num_blocks * dim * dim_wells);
    hipMalloc((void**)&d_Dnnzs, sizeof(double) * num_std_wells * dim_wells * dim_wells);
    hipMalloc((void**)&d_Bnnzs, sizeof(double) * num_blocks * dim * dim_wells);
    hipMalloc((void**)&d_Ccols, sizeof(int) * num_blocks);
    hipMalloc((void**)&d_Bcols, sizeof(int) * num_blocks);
    hipMalloc((void**)&d_val_pointers, sizeof(unsigned int) * (num_std_wells + 1));
    cudaCheckLastError("apply_gpu malloc failed");
}

// Apply the WellContributions, similar to StandardWell::apply()
// y -= (C^T *(D^-1*(   B*x)))
void WellContributionsCuda::apply(double *d_x, double *d_y)
{
    // apply MultisegmentWells

    // make sure the stream is empty if timing measurements are done
    hipStreamSynchronize(stream);

    if (num_ms_wells > 0) {
        // allocate pinned memory on host if not yet done
        if (h_x == nullptr) {
            hipHostMalloc(&h_x, sizeof(double) * N);
            hipHostMalloc(&h_y, sizeof(double) * N);
        }

        // copy vectors x and y from GPU to CPU
        hipMemcpyAsync(h_x, d_x, sizeof(double) * N, hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(h_y, d_y, sizeof(double) * N, hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        // actually apply MultisegmentWells
        for (auto& well : multisegments) {
            well->apply(h_x, h_y);
        }

        // copy vector y from CPU to GPU
        hipMemcpyAsync(d_y, h_y, sizeof(double) * N, hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream);
    }

    // apply StandardWells
    if (num_std_wells > 0) {
        int smem_size = 2 * sizeof(double) * dim_wells;
        apply_well_contributions <<< num_std_wells, 32, smem_size, stream>>>(d_Cnnzs, d_Dnnzs, d_Bnnzs, d_Ccols, d_Bcols, d_x, d_y, dim, dim_wells, d_val_pointers);
    }
}


void WellContributionsCuda::APIaddMatrix(MatrixType type, int *colIndices, double *values, unsigned int val_size)
{
    switch (type) {
    case MatrixType::C:
        hipMemcpy(d_Cnnzs + num_blocks_so_far * dim * dim_wells, values, sizeof(double) * val_size * dim * dim_wells, hipMemcpyHostToDevice);
        hipMemcpy(d_Ccols + num_blocks_so_far, colIndices, sizeof(int) * val_size, hipMemcpyHostToDevice);
        break;
    case MatrixType::D:
        hipMemcpy(d_Dnnzs + num_std_wells_so_far * dim_wells * dim_wells, values, sizeof(double) * dim_wells * dim_wells, hipMemcpyHostToDevice);
        break;
    case MatrixType::B:
        hipMemcpy(d_Bnnzs + num_blocks_so_far * dim * dim_wells, values, sizeof(double) * val_size * dim * dim_wells, hipMemcpyHostToDevice);
        hipMemcpy(d_Bcols + num_blocks_so_far, colIndices, sizeof(int) * val_size, hipMemcpyHostToDevice);
        val_pointers[num_std_wells_so_far] = num_blocks_so_far;
        if (num_std_wells_so_far == num_std_wells - 1) {
            val_pointers[num_std_wells] = num_blocks;
            hipMemcpy(d_val_pointers, val_pointers.data(), sizeof(unsigned int) * (num_std_wells + 1), hipMemcpyHostToDevice);
        }
        break;
    default:
        OPM_THROW(std::logic_error, "Error unsupported matrix ID for WellContributions::addMatrix()");
    }
    cudaCheckLastError("WellContributions::addMatrix() failed");
}

void WellContributionsCuda::setCudaStream(hipStream_t stream_)
{
    this->stream = stream_;
    for (auto& well : multisegments) {
        well->setCudaStream(stream_);
    }
}

} //namespace Opm

