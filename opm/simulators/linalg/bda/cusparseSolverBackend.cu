#include "hip/hip_runtime.h"
/*
  Copyright 2019 Equinor ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>

#include <hip/hip_runtime.h>
#include <sstream>

#include <opm/common/OpmLog/OpmLog.hpp>
#include <dune/common/timer.hh>

#include <opm/simulators/linalg/bda/cusparseSolverBackend.hpp>
#include <opm/simulators/linalg/bda/BdaResult.hpp>
#include <opm/simulators/linalg/bda/cuda_header.hpp>

#include "hipblas.h"
#include "hipsparse.h"
// For more information about cusparse, check https://docs.nvidia.com/cuda/cusparse/index.html

// iff true, the nonzeroes of the matrix are copied row-by-row into a contiguous, pinned memory array, then a single GPU memcpy is done
// otherwise, the nonzeroes of the matrix are assumed to be in a contiguous array, and a single GPU memcpy is enough
#define COPY_ROW_BY_ROW 0

namespace bda
{

using Opm::OpmLog;
using Dune::Timer;

const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
const hipsparseOperation_t operation  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
const hipsparseDirection_t order = HIPSPARSE_DIRECTION_ROW;


template <unsigned int block_size>
cusparseSolverBackend<block_size>::cusparseSolverBackend(int verbosity_, int maxit_, double tolerance_, unsigned int deviceID_) : BdaSolver<block_size>(verbosity_, maxit_, tolerance_, deviceID_) {}

template <unsigned int block_size>
cusparseSolverBackend<block_size>::~cusparseSolverBackend() {
    finalize();
}

template <unsigned int block_size>
void cusparseSolverBackend<block_size>::gpu_pbicgstab(WellContributions& wellContribs, BdaResult& res) {
    Timer t_total, t_prec(false), t_spmv(false), t_well(false), t_rest(false);
    int n = N;
    double rho = 1.0, rhop;
    double alpha, nalpha, beta;
    double omega, nomega, tmp1, tmp2;
    double norm, norm_0;
    double zero = 0.0;
    double one  = 1.0;
    double mone = -1.0;
    float it;

    if (wellContribs.getNumWells() > 0) {
        wellContribs.setCudaStream(stream);
    }

    hipsparseDbsrmv(cusparseHandle, order, operation, Nb, Nb, nnzb, &one, descr_M, d_bVals, d_bRows, d_bCols, block_size, d_x, &zero, d_r);

    hipblasDscal(cublasHandle, n, &mone, d_r, 1);
    hipblasDaxpy(cublasHandle, n, &one, d_b, 1, d_r, 1);
    hipblasDcopy(cublasHandle, n, d_r, 1, d_rw, 1);
    hipblasDcopy(cublasHandle, n, d_r, 1, d_p, 1);
    hipblasDnrm2(cublasHandle, n, d_r, 1, &norm_0);

    if (verbosity > 1) {
        std::ostringstream out;
        out << std::scientific << "cusparseSolver initial norm: " << norm_0;
        OpmLog::info(out.str());
    }

    for (it = 0.5; it < maxit; it += 0.5) {
        rhop = rho;
        hipblasDdot(cublasHandle, n, d_rw, 1, d_r, 1, &rho);

        if (it > 1) {
            beta = (rho / rhop) * (alpha / omega);
            nomega = -omega;
            hipblasDaxpy(cublasHandle, n, &nomega, d_v, 1, d_p, 1);
            hipblasDscal(cublasHandle, n, &beta, d_p, 1);
            hipblasDaxpy(cublasHandle, n, &one, d_r, 1, d_p, 1);
        }

        // apply ilu0
        hipsparseDbsrsv2_solve(cusparseHandle, order, \
                              operation, Nb, nnzb, &one, \
                              descr_L, d_mVals, d_mRows, d_mCols, block_size, info_L, d_p, d_t, policy, d_buffer);
        hipsparseDbsrsv2_solve(cusparseHandle, order, \
                              operation, Nb, nnzb, &one, \
                              descr_U, d_mVals, d_mRows, d_mCols, block_size, info_U, d_t, d_pw, policy, d_buffer);

        // spmv
        hipsparseDbsrmv(cusparseHandle, order, \
                       operation, Nb, Nb, nnzb, \
                       &one, descr_M, d_bVals, d_bRows, d_bCols, block_size, d_pw, &zero, d_v);

        // apply wellContributions
        if (wellContribs.getNumWells() > 0) {
            wellContribs.apply(d_pw, d_v);
        }

        hipblasDdot(cublasHandle, n, d_rw, 1, d_v, 1, &tmp1);
        alpha = rho / tmp1;
        nalpha = -alpha;
        hipblasDaxpy(cublasHandle, n, &nalpha, d_v, 1, d_r, 1);
        hipblasDaxpy(cublasHandle, n, &alpha, d_pw, 1, d_x, 1);
        hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);

        if (norm < tolerance * norm_0) {
            break;
        }

        it += 0.5;

        // apply ilu0
        hipsparseDbsrsv2_solve(cusparseHandle, order, \
                              operation, Nb, nnzb, &one, \
                              descr_L, d_mVals, d_mRows, d_mCols, block_size, info_L, d_r, d_t, policy, d_buffer);
        hipsparseDbsrsv2_solve(cusparseHandle, order, \
                              operation, Nb, nnzb, &one, \
                              descr_U, d_mVals, d_mRows, d_mCols, block_size, info_U, d_t, d_s, policy, d_buffer);

        // spmv
        hipsparseDbsrmv(cusparseHandle, order, \
                       operation, Nb, Nb, nnzb, &one, descr_M, \
                       d_bVals, d_bRows, d_bCols, block_size, d_s, &zero, d_t);

        // apply wellContributions
        if (wellContribs.getNumWells() > 0) {
            wellContribs.apply(d_s, d_t);
        }

        hipblasDdot(cublasHandle, n, d_t, 1, d_r, 1, &tmp1);
        hipblasDdot(cublasHandle, n, d_t, 1, d_t, 1, &tmp2);
        omega = tmp1 / tmp2;
        nomega = -omega;
        hipblasDaxpy(cublasHandle, n, &omega, d_s, 1, d_x, 1);
        hipblasDaxpy(cublasHandle, n, &nomega, d_t, 1, d_r, 1);

        hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);


        if (norm < tolerance * norm_0) {
            break;
        }

        if (verbosity > 1) {
            std::ostringstream out;
            out << "it: " << it << std::scientific << ", norm: " << norm;
            OpmLog::info(out.str());
        }
    }

    res.iterations = std::min(it, (float)maxit);
    res.reduction = norm / norm_0;
    res.conv_rate  = static_cast<double>(pow(res.reduction, 1.0 / it));
    res.elapsed = t_total.stop();
    res.converged = (it != (maxit + 0.5));

    if (verbosity > 0) {
        std::ostringstream out;
        out << "=== converged: " << res.converged << ", conv_rate: " << res.conv_rate << ", time: " << res.elapsed << \
            ", time per iteration: " << res.elapsed / it << ", iterations: " << it;
        OpmLog::info(out.str());
    }
}


template <unsigned int block_size>
void cusparseSolverBackend<block_size>::initialize(int N, int nnz, int dim) {
    this->N = N;
    this->nnz = nnz;
    this->nnzb = nnz / block_size / block_size;
    Nb = (N + dim - 1) / dim;
    std::ostringstream out;
    out << "Initializing GPU, matrix size: " << N << " blocks, nnz: " << nnzb << " blocks";
    OpmLog::info(out.str());
    out.str("");
    out.clear();
    out << "Maxit: " << maxit << std::scientific << ", tolerance: " << tolerance;
    OpmLog::info(out.str());

    hipSetDevice(deviceID);
    cudaCheckLastError("Could not get device");
    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceID);
    cudaCheckLastError("Could not get device properties");
    out.str("");
    out.clear();
    out << "Name GPU: " << props.name << ", Compute Capability: " << props.major << "." << props.minor;
    OpmLog::info(out.str());

    hipStreamCreate(&stream);
    cudaCheckLastError("Could not create stream");

    hipblasCreate(&cublasHandle);
    cudaCheckLastError("Could not create cublasHandle");

    hipsparseCreate(&cusparseHandle);
    cudaCheckLastError("Could not create cusparseHandle");

    hipMalloc((void**)&d_x, sizeof(double) * N);
    hipMalloc((void**)&d_b, sizeof(double) * N);
    hipMalloc((void**)&d_r, sizeof(double) * N);
    hipMalloc((void**)&d_rw, sizeof(double) * N);
    hipMalloc((void**)&d_p, sizeof(double) * N);
    hipMalloc((void**)&d_pw, sizeof(double) * N);
    hipMalloc((void**)&d_s, sizeof(double) * N);
    hipMalloc((void**)&d_t, sizeof(double) * N);
    hipMalloc((void**)&d_v, sizeof(double) * N);
    hipMalloc((void**)&d_bVals, sizeof(double) * nnz);
    hipMalloc((void**)&d_bCols, sizeof(int) * nnzb);
    hipMalloc((void**)&d_bRows, sizeof(int) * (Nb + 1));
    hipMalloc((void**)&d_mVals, sizeof(double) * nnz);
    cudaCheckLastError("Could not allocate enough memory on GPU");

    hipblasSetStream(cublasHandle, stream);
    cudaCheckLastError("Could not set stream to cublas");
    hipsparseSetStream(cusparseHandle, stream);
    cudaCheckLastError("Could not set stream to cusparse");

#if COPY_ROW_BY_ROW
    hipHostMalloc((void**)&vals_contiguous, sizeof(double) * nnz);
    cudaCheckLastError("Could not allocate pinned memory");
#endif

    initialized = true;
} // end initialize()

template <unsigned int block_size>
void cusparseSolverBackend<block_size>::finalize() {
    if (initialized) {
        hipFree(d_x);
        hipFree(d_b);
        hipFree(d_r);
        hipFree(d_rw);
        hipFree(d_p);
        hipFree(d_pw);
        hipFree(d_s);
        hipFree(d_t);
        hipFree(d_v);
        hipFree(d_mVals);
        hipFree(d_bVals);
        hipFree(d_bCols);
        hipFree(d_bRows);
        hipFree(d_buffer);
        hipsparseDestroyBsrilu02Info(info_M);
        hipsparseDestroyBsrsv2Info(info_L);
        hipsparseDestroyBsrsv2Info(info_U);
        hipsparseDestroyMatDescr(descr_B);
        hipsparseDestroyMatDescr(descr_M);
        hipsparseDestroyMatDescr(descr_L);
        hipsparseDestroyMatDescr(descr_U);
        hipsparseDestroy(cusparseHandle);
        hipblasDestroy(cublasHandle);
#if COPY_ROW_BY_ROW
        hipHostFree(vals_contiguous);
#endif
        hipStreamDestroy(stream);
    }
} // end finalize()


template <unsigned int block_size>
void cusparseSolverBackend<block_size>::copy_system_to_gpu(double *vals, int *rows, int *cols, double *b) {
    Timer t;

#if COPY_ROW_BY_ROW
    int sum = 0;
    for (int i = 0; i < Nb; ++i) {
        int size_row = rows[i + 1] - rows[i];
        memcpy(vals_contiguous + sum, vals + sum, size_row * sizeof(double) * block_size * block_size);
        sum += size_row * block_size * block_size;
    }
    hipMemcpyAsync(d_bVals, vals_contiguous, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#else
    hipMemcpyAsync(d_bVals, vals, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#endif

    hipMemcpyAsync(d_bCols, cols, nnzb * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_bRows, rows, (Nb + 1) * sizeof(int), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b, N * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemsetAsync(d_x, 0, sizeof(double) * N, stream);

    if (verbosity > 2) {
        hipStreamSynchronize(stream);
        std::ostringstream out;
        out << "cusparseSolver::copy_system_to_gpu(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }
} // end copy_system_to_gpu()


// don't copy rowpointers and colindices, they stay the same
template <unsigned int block_size>
void cusparseSolverBackend<block_size>::update_system_on_gpu(double *vals, int *rows, double *b) {
    Timer t;

#if COPY_ROW_BY_ROW
    int sum = 0;
    for (int i = 0; i < Nb; ++i) {
        int size_row = rows[i + 1] - rows[i];
        memcpy(vals_contiguous + sum, vals + sum, size_row * sizeof(double) * block_size * block_size);
        sum += size_row * block_size * block_size;
    }
    hipMemcpyAsync(d_bVals, vals_contiguous, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#else
    hipMemcpyAsync(d_bVals, vals, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#endif

    hipMemcpyAsync(d_b, b, N * sizeof(double), hipMemcpyHostToDevice, stream);
    hipMemsetAsync(d_x, 0, sizeof(double) * N, stream);

    if (verbosity > 2) {
        hipStreamSynchronize(stream);
        std::ostringstream out;
        out << "cusparseSolver::update_system_on_gpu(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }
} // end update_system_on_gpu()


template <unsigned int block_size>
void cusparseSolverBackend<block_size>::reset_prec_on_gpu() {
    hipMemcpyAsync(d_mVals, d_bVals, nnz  * sizeof(double), hipMemcpyDeviceToDevice, stream);
}


template <unsigned int block_size>
bool cusparseSolverBackend<block_size>::analyse_matrix() {

    int d_bufferSize_M, d_bufferSize_L, d_bufferSize_U, d_bufferSize;
    Timer t;

    hipsparseCreateMatDescr(&descr_B);
    hipsparseCreateMatDescr(&descr_M);
    hipsparseSetMatType(descr_B, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
    const hipsparseIndexBase_t base_type = HIPSPARSE_INDEX_BASE_ZERO;     // matrices from Flow are base0

    hipsparseSetMatIndexBase(descr_B, base_type);
    hipsparseSetMatIndexBase(descr_M, base_type);

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, base_type);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatIndexBase(descr_U, base_type);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
    cudaCheckLastError("Could not initialize matrix descriptions");

    hipsparseCreateBsrilu02Info(&info_M);
    hipsparseCreateBsrsv2Info(&info_L);
    hipsparseCreateBsrsv2Info(&info_U);
    cudaCheckLastError("Could not create analysis info");

    hipsparseDbsrilu02_bufferSize(cusparseHandle, order, Nb, nnzb,
                                 descr_M, d_bVals, d_bRows, d_bCols, block_size, info_M, &d_bufferSize_M);
    hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzb,
                               descr_L, d_bVals, d_bRows, d_bCols, block_size, info_L, &d_bufferSize_L);
    hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzb,
                               descr_U, d_bVals, d_bRows, d_bCols, block_size, info_U, &d_bufferSize_U);
    cudaCheckLastError();
    d_bufferSize = std::max(d_bufferSize_M, std::max(d_bufferSize_L, d_bufferSize_U));

    hipMalloc((void**)&d_buffer, d_bufferSize);

    // analysis of ilu LU decomposition
    hipsparseDbsrilu02_analysis(cusparseHandle, order, \
                               Nb, nnzb, descr_B, d_bVals, d_bRows, d_bCols, \
                               block_size, info_M, policy, d_buffer);

    int structural_zero;
    hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        return false;
    }

    // analysis of ilu apply
    hipsparseDbsrsv2_analysis(cusparseHandle, order, operation, \
                             Nb, nnzb, descr_L, d_bVals, d_bRows, d_bCols, \
                             block_size, info_L, policy, d_buffer);

    hipsparseDbsrsv2_analysis(cusparseHandle, order, operation, \
                             Nb, nnzb, descr_U, d_bVals, d_bRows, d_bCols, \
                             block_size, info_U, policy, d_buffer);
    cudaCheckLastError("Could not analyse level information");

    if (verbosity > 2) {
        hipStreamSynchronize(stream);
        std::ostringstream out;
        out << "cusparseSolver::analyse_matrix(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }

    analysis_done = true;

    return true;
} // end analyse_matrix()

template <unsigned int block_size>
bool cusparseSolverBackend<block_size>::create_preconditioner() {
    Timer t;

    d_mCols = d_bCols;
    d_mRows = d_bRows;
    hipsparseDbsrilu02(cusparseHandle, order, \
                      Nb, nnzb, descr_M, d_mVals, d_mRows, d_mCols, \
                      block_size, info_M, policy, d_buffer);
    cudaCheckLastError("Could not perform ilu decomposition");

    int structural_zero;
    // hipsparseXbsrilu02_zeroPivot() calls hipDeviceSynchronize()
    hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        return false;
    }

    if (verbosity > 2) {
        hipStreamSynchronize(stream);
        std::ostringstream out;
        out << "cusparseSolver::create_preconditioner(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }
    return true;
} // end create_preconditioner()


template <unsigned int block_size>
void cusparseSolverBackend<block_size>::solve_system(WellContributions& wellContribs, BdaResult &res) {
    // actually solve
    gpu_pbicgstab(wellContribs, res);
    hipStreamSynchronize(stream);
    cudaCheckLastError("Something went wrong during the GPU solve");
} // end solve_system()


// copy result to host memory
// caller must be sure that x is a valid array
template <unsigned int block_size>
void cusparseSolverBackend<block_size>::get_result(double *x) {
    Timer t;

    hipMemcpyAsync(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    if (verbosity > 2) {
        std::ostringstream out;
        out << "cusparseSolver::get_result(): " << t.stop() << " s";
        OpmLog::info(out.str());
    }
} // end get_result()



template <unsigned int block_size>
SolverStatus cusparseSolverBackend<block_size>::solve_system(int N, int nnz, int dim, double *vals, int *rows, int *cols, double *b, WellContributions& wellContribs, BdaResult &res) {
    if (initialized == false) {
        initialize(N, nnz, dim);
        copy_system_to_gpu(vals, rows, cols, b);
    } else {
        update_system_on_gpu(vals, rows, b);
    }
    if (analysis_done == false) {
        if (!analyse_matrix()) {
            return SolverStatus::BDA_SOLVER_ANALYSIS_FAILED;
        }
    }
    reset_prec_on_gpu();
    if (create_preconditioner()) {
        solve_system(wellContribs, res);
    } else {
        return SolverStatus::BDA_SOLVER_CREATE_PRECONDITIONER_FAILED;
    }
    return SolverStatus::BDA_SOLVER_SUCCESS;
}


#define INSTANTIATE_BDA_FUNCTIONS(n)                                                       \
template cusparseSolverBackend<n>::cusparseSolverBackend(int, int, double, unsigned int);  \

INSTANTIATE_BDA_FUNCTIONS(1);
INSTANTIATE_BDA_FUNCTIONS(2);
INSTANTIATE_BDA_FUNCTIONS(3);
INSTANTIATE_BDA_FUNCTIONS(4);
INSTANTIATE_BDA_FUNCTIONS(5);
INSTANTIATE_BDA_FUNCTIONS(6);

#undef INSTANTIATE_BDA_FUNCTIONS

} // namespace bda


