/*
  Copyright 2021 Equinor ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <config.h>
#include <sstream>

#include <opm/common/OpmLog/OpmLog.hpp>

#include <amgcl/backend/cuda.hpp>
#include <amgcl/relaxation/cusparse_ilu0.hpp>
#include <opm/simulators/linalg/bda/amgclSolverBackend.hpp>

/// This file is only compiled when both amgcl and CUDA are found by CMake

namespace bda
{

using Opm::OpmLog;


template <unsigned int block_size>
void amgclSolverBackend<block_size>::solve_cuda(double *b) {
    typedef amgcl::backend::cuda<double> CUDA_Backend;
    typedef amgcl::make_solver<amgcl::runtime::preconditioner<CUDA_Backend>, amgcl::runtime::solver::wrapper<CUDA_Backend> > CUDA_Solver;

    static typename CUDA_Backend::params CUDA_bprm; // amgcl backend parameters, only used for cusparseHandle

    // initialize cusparse handle for amgcl, cannot merge this call_once with 'print solver structure'
    std::call_once(cuda_initialize, [&](){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, deviceID);
        std::ostringstream out;
        out << prop.name << std::endl;
        OpmLog::info(out.str());
        hipsparseCreate(&CUDA_bprm.cusparse_handle);
    });

    // create matrix object
    auto A = std::tie(N, A_rows, A_cols, A_vals);

    // create solver and construct preconditioner
    // don't reuse this unless the preconditioner can be reused
    CUDA_Solver solve(A, prm, CUDA_bprm);

    // print solver structure (once)
    std::call_once(print_info, [&](){
        std::ostringstream out;
        out << solve << std::endl;
        OpmLog::info(out.str());
    });

    thrust::device_vector<double> B(b, b + N);
    thrust::device_vector<double> X(N, 0.0);

    // actually solve
    std::tie(iters, error) = solve(B, X);

    thrust::copy(X.begin(), X.end(), x.begin());
}


#define INSTANTIATE_BDA_FUNCTIONS(n)                      \
template void amgclSolverBackend<n>::solve_cuda(double*); \

INSTANTIATE_BDA_FUNCTIONS(1);
INSTANTIATE_BDA_FUNCTIONS(2);
INSTANTIATE_BDA_FUNCTIONS(3);
INSTANTIATE_BDA_FUNCTIONS(4);
INSTANTIATE_BDA_FUNCTIONS(5);
INSTANTIATE_BDA_FUNCTIONS(6);

#undef INSTANTIATE_BDA_FUNCTIONS

} // namespace bda

